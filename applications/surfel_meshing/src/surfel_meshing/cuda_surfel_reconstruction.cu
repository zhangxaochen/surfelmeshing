#include "hip/hip_runtime.h"
// Copyright 2018 ETH Zürich, Thomas Schöps
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its contributors
//    may be used to endorse or promote products derived from this software
//    without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.


// Avoid warnings in Qt includes with CUDA compiler
#pragma GCC diagnostic ignored "-Wattributes"
// Avoid warnings in Eigen includes with CUDA compiler
#pragma diag_suppress code_is_unreachable

#include "surfel_meshing/cuda_surfel_reconstruction.cuh"

#include <cub/block/block_reduce.cuh>
#include <cub/device/device_scan.cuh>
#include <libvis/point_cloud.h>
#include <hip/hip_math_constants.h>

#include "surfel_meshing/cuda_matrix.cuh"
#include "surfel_meshing/cuda_util.cuh"
#include "surfel_meshing/surfel.h"

// Uncomment this to run CUDA kernels sequentially for debugging.
// #define CUDA_SEQUENTIAL_CHECKS

namespace vis {

// This threshold is not exposed as a program argument since I am not sure
// whether any other value than 0 would be useful.
constexpr float kSurfelNormalToViewingDirThreshold = 0;

// For a surfel with a given radius, the observation radius can be up to this
// factor worse (larger) while the observation is still integrated into the
// surfel. Observations with larger radii than that are discarded.
// TODO: Expose as a program argument?
constexpr float kMaxObservationRadiusFactorForIntegration = 1.5f;

// Not exposed as a program argument since it did not seem to work well.
constexpr bool kCheckScaleCompatibilityForIntegration = false;

// Not exposed as a program argument since disabling it might not make sense.
constexpr bool kCheckScaleCompatibilityForNeighborAssignment = true;

// If this is set to true, slightly occluded surfels will be protected better,
// but the surfel integration will be unable to merge duplicate surfaces after
// loop closures.
constexpr bool kProtectSlightlyOccludedSurfels = false;
constexpr float kOcclusionDepthFactor = 0.01f;


__forceinline__ __device__ bool IsSurfelActiveForIntegration(
    u32 surfel_index,
    const CUDABuffer_<float>& surfels,
    u32 frame_index,
    int surfel_integration_active_window_size) {
  // Alternatives:
  // kSurfelCreationStamp --> surfels are always deactivated after a certain time and never reactivated. Creates the least artifacts during deformations, but leads to many surfels.
  // kSurfelLastUpdateStamp --> surfels stay active. Leads to problems during deformation at observation boundaries (where the surfels are next to each other, but kSurfelLastUpdateStamp differs strongly).
  return static_cast<int>(*reinterpret_cast<const u32*>(&surfels(kSurfelLastUpdateStamp, surfel_index))) >
             static_cast<int>(frame_index) - surfel_integration_active_window_size;
}


__global__ void CreateNewSurfelsCUDASerializingKernel(
    CUDABuffer_<u16> depth_buffer,
    CUDABuffer_<u32> supporting_surfels,
    CUDABuffer_<u32> conflicting_surfels,
    CUDABuffer_<u8> new_surfel_flag_vector) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x < depth_buffer.width() && y < depth_buffer.height()) {
    // TODO: Is this border necessary here, or should it rather be integrated into the depth map erosion?
    constexpr int kBorder = 1;
    bool new_surfel = x >= kBorder &&
                      y >= kBorder &&
                      x < depth_buffer.width() - kBorder &&
                      y < depth_buffer.height() - kBorder &&
                      depth_buffer(y, x) > 0 &&
                      supporting_surfels(y, x) == Surfel::kInvalidIndex &&
                      conflicting_surfels(y, x) == Surfel::kInvalidIndex;
    u32 seq_index = x + y * depth_buffer.width();
    new_surfel_flag_vector(0, seq_index) = new_surfel ? 1 : 0;
  }
}

__global__ void CreateNewSurfelsCUDACreationKernel(
    u32 frame_index,
    float inv_depth_scaling,
    float fx_inv, float fy_inv, float cx_inv, float cy_inv,
    CUDAMatrix3x4 global_T_local,
    CUDABuffer_<u16> depth_buffer,
    CUDABuffer_<float2> normals_buffer,
    CUDABuffer_<float> radius_buffer,
    CUDABuffer_<uchar3> color_buffer,
    CUDABuffer_<u32> supporting_surfels,
    CUDABuffer_<u8> new_surfel_flag_vector,
    CUDABuffer_<u32> new_surfel_indices,
    u32 surfel_count,
    CUDABuffer_<float> surfels,
    float radius_factor_for_regularization_neighbors_squared) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (x < depth_buffer.width() && y < depth_buffer.height()) {
    u32 seq_index = x + y * depth_buffer.width();
    if (new_surfel_flag_vector(0, seq_index) != 1) {
      return;
    }
    
    u32 surfel_index = surfel_count + new_surfel_indices(0, seq_index);
    
    float depth = inv_depth_scaling * depth_buffer(y, x);
    float3 local_position;
    UnprojectPoint(x, y, depth, fx_inv, fy_inv, cx_inv, cy_inv, &local_position);
    float3 global_position = global_T_local * local_position;
    
    surfels(kSurfelX, surfel_index) = global_position.x;
    surfels(kSurfelY, surfel_index) = global_position.y;
    surfels(kSurfelZ, surfel_index) = global_position.z;
    surfels(kSurfelSmoothX, surfel_index) = global_position.x;
    surfels(kSurfelSmoothY, surfel_index) = global_position.y;
    surfels(kSurfelSmoothZ, surfel_index) = global_position.z;
    
    float2 normal_xy = normals_buffer(y, x);
    const float normal_z = -sqrtf(::max(0.f, 1 - normal_xy.x * normal_xy.x - normal_xy.y * normal_xy.y));
    float3 global_normal = global_T_local.Rotate(make_float3(normal_xy.x, normal_xy.y, normal_z));
    
    surfels(kSurfelNormalX, surfel_index) = global_normal.x;
    surfels(kSurfelNormalY, surfel_index) = global_normal.y;
    surfels(kSurfelNormalZ, surfel_index) = global_normal.z;
    
    uchar3 color = color_buffer(y, x);
    *(reinterpret_cast<uchar4*>(&surfels(kSurfelColor, surfel_index))) = make_uchar4(color.x, color.y, color.z, 0);
    
    surfels(kSurfelConfidence, surfel_index) = 1;
    *reinterpret_cast<u32*>(&surfels(kSurfelCreationStamp, surfel_index)) = frame_index;
    *reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, surfel_index)) = frame_index;
    
    const float radius_squared = radius_buffer(y, x);
    surfels(kSurfelRadiusSquared, surfel_index) = radius_squared;
    
    // Determine initial neighbors.
    float3 neighbor_position_sum = make_float3(0, 0, 0);
    int existing_neighbor_count_plus_1 = 1;
    constexpr int kDirectionsX[4] = {-1, 1, 0, 0};
    constexpr int kDirectionsY[4] = {0, 0, -1, 1};
    for (int direction = 0; direction < 4; ++ direction) {
      u32 neighbor_index = supporting_surfels(y + kDirectionsY[direction], x + kDirectionsX[direction]);

      if (neighbor_index != Surfel::kInvalidIndex) {
        float3 this_to_neighbor = make_float3(surfels(kSurfelX, neighbor_index) - global_position.x,
                                              surfels(kSurfelY, neighbor_index) - global_position.y,
                                              surfels(kSurfelZ, neighbor_index) - global_position.z);
        float distance_squared =
            this_to_neighbor.x * this_to_neighbor.x + this_to_neighbor.y * this_to_neighbor.y + this_to_neighbor.z * this_to_neighbor.z;
        if (distance_squared > radius_factor_for_regularization_neighbors_squared * radius_squared) {
          neighbor_index = Surfel::kInvalidIndex;
        } else {
          neighbor_position_sum = make_float3(
              neighbor_position_sum.x + surfels(kSurfelSmoothX, neighbor_index),
              neighbor_position_sum.y + surfels(kSurfelSmoothY, neighbor_index),
              neighbor_position_sum.z + surfels(kSurfelSmoothZ, neighbor_index));
          ++ existing_neighbor_count_plus_1;
        }
      } else {
        u32 seq_neighbor_index = (x + kDirectionsX[direction]) + (y + kDirectionsY[direction]) * depth_buffer.width();
        if (new_surfel_flag_vector(0, seq_neighbor_index) == 1) {
          float other_depth = inv_depth_scaling * depth_buffer(y + kDirectionsY[direction], x + kDirectionsX[direction]);
          float approximate_distance_squared = (depth - other_depth) * (depth - other_depth);
          if (approximate_distance_squared <= radius_factor_for_regularization_neighbors_squared * radius_squared) {
            neighbor_index = surfel_count + new_surfel_indices(0, seq_neighbor_index);
          }
        }
      }
      
      *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + direction, surfel_index)) = neighbor_index;
    }
    
    // Try to get a better initialization for the regularized surfel position.
    surfels(kSurfelSmoothX, surfel_index) = (surfels(kSurfelSmoothX, surfel_index) + neighbor_position_sum.x) / existing_neighbor_count_plus_1;
    surfels(kSurfelSmoothY, surfel_index) = (surfels(kSurfelSmoothY, surfel_index) + neighbor_position_sum.y) / existing_neighbor_count_plus_1;
    surfels(kSurfelSmoothZ, surfel_index) = (surfels(kSurfelSmoothZ, surfel_index) + neighbor_position_sum.z) / existing_neighbor_count_plus_1;
  }
}

void CreateNewSurfelsCUDA(
    hipStream_t stream,
    u32 frame_index,
    const SE3f& global_T_local,
    float depth_scaling,
    float radius_factor_for_regularization_neighbors,
    const PinholeCamera4f& depth_camera,
    const CUDABuffer<u16>& depth_buffer,
    const CUDABuffer<float2>& normals_buffer,
    const CUDABuffer<float>& radius_buffer,
    const CUDABuffer<Vec3u8>& color_buffer,
    const CUDABuffer<u32>& supporting_surfels,
    const CUDABuffer<u32>& conflicting_surfels,
    void** new_surfels_temp_storage,
    usize* new_surfels_temp_storage_bytes,
    CUDABuffer<u8>* new_surfel_flag_vector,
    CUDABuffer<u32>* new_surfel_indices,
    u32 surfel_count,
    CUDABuffer<float>* surfels,
    u32* new_surfel_count,
    u8* new_surfel_count_2) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  const float fx = depth_camera.parameters()[0];
  const float fy = depth_camera.parameters()[1];
  const float cx = depth_camera.parameters()[2];
  const float cy = depth_camera.parameters()[3];
  
  // Unprojection intrinsics for pixel center convention.
  const float fx_inv = 1.0f / fx;
  const float fy_inv = 1.0f / fy;
  const float cx_pixel_center = cx - 0.5f;
  const float cy_pixel_center = cy - 0.5f;
  const float cx_inv_pixel_center = -cx_pixel_center / fx;
  const float cy_inv_pixel_center = -cy_pixel_center / fy;
  
  // The first kernel marks in a sequential (non-pitched) vector whether a new surfel is created for the corresponding pixel or not.
  constexpr int kBlockWidth = 32;
  constexpr int kBlockHeight = 32;
  dim3 grid_dim(GetBlockCount(depth_buffer.width(), kBlockWidth),
                GetBlockCount(depth_buffer.height(), kBlockHeight));
  dim3 block_dim(kBlockWidth, kBlockHeight);
  
  CreateNewSurfelsCUDASerializingKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      depth_buffer.ToCUDA(),
      supporting_surfels.ToCUDA(),
      conflicting_surfels.ToCUDA(),
      new_surfel_flag_vector->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  // Indices for the new surfels are computed with a parallel exclusive prefix sum from CUB.
  if (*new_surfels_temp_storage_bytes == 0) {
    hipcub::DeviceScan::ExclusiveSum(
        *new_surfels_temp_storage,
        *new_surfels_temp_storage_bytes,
        new_surfel_flag_vector->ToCUDA().address(),
        new_surfel_indices->ToCUDA().address(),
        depth_buffer.width() * depth_buffer.height(),
        stream);
    
    hipMalloc(new_surfels_temp_storage, *new_surfels_temp_storage_bytes);
  }
  
  hipcub::DeviceScan::ExclusiveSum(
      *new_surfels_temp_storage,
      *new_surfels_temp_storage_bytes,
      new_surfel_flag_vector->ToCUDA().address(),
      new_surfel_indices->ToCUDA().address(),
      depth_buffer.width() * depth_buffer.height(),
      stream);
  
  // Read back the number of new surfels to the CPU by reading the last element
  // in new_surfel_indices and new_surfel_flag_vector.
  // TODO: Do this concurrently with the next kernel call?
  new_surfel_indices->DownloadPartAsync(
      (depth_buffer.width() * depth_buffer.height() - 1) * sizeof(u32),
      1 * sizeof(u32),
      stream,
      new_surfel_count);
  new_surfel_flag_vector->DownloadPartAsync(
      (depth_buffer.width() * depth_buffer.height() - 1) * sizeof(u8),
      1 * sizeof(u8),
      stream,
      new_surfel_count_2);
  
  // Now that the indices are known, the actual surfel creation is done.
  CreateNewSurfelsCUDACreationKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      1.0f / depth_scaling,
      fx_inv, fy_inv, cx_inv_pixel_center, cy_inv_pixel_center,
      CUDAMatrix3x4(global_T_local.matrix3x4()),
      depth_buffer.ToCUDA(),
      normals_buffer.ToCUDA(),
      radius_buffer.ToCUDA(),
      *reinterpret_cast<const CUDABuffer_<uchar3>*>(&color_buffer.ToCUDA()),
      supporting_surfels.ToCUDA(),
      new_surfel_flag_vector->ToCUDA(),
      new_surfel_indices->ToCUDA(),
      surfel_count,
      surfels->ToCUDA(),
      radius_factor_for_regularization_neighbors * radius_factor_for_regularization_neighbors);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
}


template <bool visualize_last_update_timestamp,
          bool visualize_creation_timestamp,
          bool visualize_radii,
          bool visualize_normals>
__global__ void UpdateSurfelVertexBufferCUDAKernel(
    u32 frame_index,
    int surfel_integration_active_window_size,
    u32 point_size_in_floats,
    u32 surfel_count,
    CUDABuffer_<float> surfels,
    u32 latest_triangulated_frame_index,
    u32 latest_mesh_surfel_count,
    float* vertex_buffer_ptr) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    const u32 surfel_creation_stamp = *reinterpret_cast<u32*>(&surfels(kSurfelCreationStamp, surfel_index));
    // Only output if it is an old surfel that has not been replaced since the last mesh was created,
    // or if it is a new surfel which does not appear in the mesh yet.
    const bool output_vertex = surfel_creation_stamp <= latest_triangulated_frame_index ||
                               surfel_index >= latest_mesh_surfel_count;
    
    // Vertex layout (Point3fC3u8):
    // float x, float y, float z, u8 r, u8 g, u8 b, u8 unused;
    // Using NaN for one of the vertex coordinates to prevent it from being
    // drawn if the surfel was replaced recently and the triangulation not
    // adjusted yet. This makes the adjacent triangles disappear. Not sure
    // whether that is portable, but it works as intended on my system ...
    vertex_buffer_ptr[surfel_index * point_size_in_floats + 0] = output_vertex ? surfels(kSurfelSmoothX, surfel_index) : HIP_NAN_F;
    vertex_buffer_ptr[surfel_index * point_size_in_floats + 1] = surfels(kSurfelSmoothY, surfel_index);
    vertex_buffer_ptr[surfel_index * point_size_in_floats + 2] = surfels(kSurfelSmoothZ, surfel_index);
    
    if (visualize_last_update_timestamp || visualize_creation_timestamp) {
      const u32 last_update_timestamp = *reinterpret_cast<u32*>(&surfels(visualize_creation_timestamp ? kSurfelCreationStamp : kSurfelLastUpdateStamp, surfel_index));
      const int age = frame_index - last_update_timestamp;
      constexpr int kVisualizationMinAge = 1;
      const int kVisualizationMaxAge = visualize_creation_timestamp ? 3000 : surfel_integration_active_window_size;
      if (age < kVisualizationMinAge) {
        // Special color for surfels updated in the last frame: red.
        uchar4 color = make_uchar4(255, 80, 80, 0);
        vertex_buffer_ptr[surfel_index * point_size_in_floats + 3] = *reinterpret_cast<float*>(&color);
      } else if (age > kVisualizationMaxAge) {
        // Old surfels: blue
        uchar4 color = make_uchar4(40, 40, 255, 0);
        vertex_buffer_ptr[surfel_index * point_size_in_floats + 3] = *reinterpret_cast<float*>(&color);
      } else {
        float blend_factor = (age - kVisualizationMinAge) * 1.0f / (kVisualizationMaxAge - kVisualizationMinAge);
        blend_factor = ::min(1.0f, ::max(0.0f, blend_factor));
        u8 intensity = 255 - static_cast<u8>(255.99f * blend_factor);
        uchar4 color = make_uchar4(intensity, intensity, intensity, 0);
        vertex_buffer_ptr[surfel_index * point_size_in_floats + 3] = *reinterpret_cast<float*>(&color);
      }
    } else if (visualize_radii) {
      const float radius_squared = surfels(kSurfelRadiusSquared, surfel_index);
      const float radius = sqrtf(radius_squared);
      constexpr float kVisualizationMinRadius = 0.0005f;  // 0.5 mm
      constexpr float kVisualizationMaxRadius = 0.01f;   // 1 cm
      float blend_factor = (radius - kVisualizationMinRadius) / (kVisualizationMaxRadius - kVisualizationMinRadius);
      blend_factor = ::min(1.0f, ::max(0.0f, blend_factor));
      u8 red = 255.99f * blend_factor;
      u8 green = 255 - red;
      u8 blue = 80;
      uchar4 color = make_uchar4(red, green, blue, 0);
      vertex_buffer_ptr[surfel_index * point_size_in_floats + 3] = *reinterpret_cast<float*>(&color);
    } else if (visualize_normals) {
      float3 normal = make_float3(surfels(kSurfelNormalX, surfel_index),
                                  surfels(kSurfelNormalY, surfel_index),
                                  surfels(kSurfelNormalZ, surfel_index));
      uchar4 color = make_uchar4(255.99f / 2.0f * (normal.x + 1.0f),
                                 255.99f / 2.0f * (normal.y + 1.0f),
                                 255.99f / 2.0f * (normal.z + 1.0f),
                                 0);
      vertex_buffer_ptr[surfel_index * point_size_in_floats + 3] = *reinterpret_cast<float*>(&color);
    } else {
      vertex_buffer_ptr[surfel_index * point_size_in_floats + 3] = surfels(kSurfelColor, surfel_index);
    }
  }
}

void UpdateSurfelVertexBufferCUDA(
    hipStream_t stream,
    u32 frame_index,
    int surfel_integration_active_window_size,
    u32 surfel_count,
    const CUDABuffer<float>& surfels,
    u32 latest_triangulated_frame_index,
    u32 latest_mesh_surfel_count,
    hipGraphicsResource_t vertex_buffer_resource,
    bool visualize_last_update_timestamp,
    bool visualize_creation_timestamp,
    bool visualize_radii,
    bool visualize_normals) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  // Map OpenGL buffer object for writing from CUDA.
  hipGraphicsMapResources(1, &vertex_buffer_resource, stream);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  usize num_bytes;
  float* vertex_buffer_ptr;
  hipGraphicsResourceGetMappedPointer((void**)&vertex_buffer_ptr, &num_bytes, vertex_buffer_resource);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  constexpr int kBlockWidth = 1024;
  dim3 grid_dim(GetBlockCount(surfel_count, kBlockWidth));
  dim3 block_dim(kBlockWidth);
  
  CHECK(sizeof(Point3fC3u8) % sizeof(float) == 0);
  u32 point_size_in_floats = sizeof(Point3fC3u8) / sizeof(float);
  
  #define CALL_KERNEL(visualize_last_update_timestamp, \
                      visualize_creation_timestamp, \
                      visualize_radii, \
                      visualize_normals) \
      UpdateSurfelVertexBufferCUDAKernel \
      <visualize_last_update_timestamp, \
       visualize_creation_timestamp, \
       visualize_radii, \
       visualize_normals> \
      <<<grid_dim, block_dim, 0, stream>>>( \
          frame_index, \
          surfel_integration_active_window_size, \
          point_size_in_floats, \
          surfel_count, \
          surfels.ToCUDA(), \
          latest_triangulated_frame_index, \
          latest_mesh_surfel_count, \
          vertex_buffer_ptr)
  if (visualize_last_update_timestamp) {
    CALL_KERNEL(true, false, false, false);
  } else if (visualize_creation_timestamp) {
    CALL_KERNEL(false, true, false, false);
  } else if (visualize_radii) {
    CALL_KERNEL(false, false, true, false);
  } else if (visualize_normals) {
    CALL_KERNEL(false, false, false, true);
  }else {
    CALL_KERNEL(false, false, false, false);
  }
  #undef CALL_KERNEL
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  hipGraphicsUnmapResources(1, &vertex_buffer_resource, stream);
}


__global__ void UpdateNeighborIndexBufferCUDAKernel(
    u32 surfel_count,
    CUDABuffer_<float> surfels,
    unsigned int* neighbor_index_buffer_ptr) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    #pragma unroll
    for (int i = 0; i < kSurfelNeighborCount; ++ i) {
      neighbor_index_buffer_ptr[surfel_index * 2 * kSurfelNeighborCount + 2 * i + 0] = surfel_index;
      u32 neighbor_index = *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + i, surfel_index));
      neighbor_index_buffer_ptr[surfel_index * 2 * kSurfelNeighborCount + 2 * i + 1] =
          (neighbor_index == Surfel::kInvalidIndex) ? surfel_index : neighbor_index;
    }
  }
}

void UpdateNeighborIndexBufferCUDA(
    hipStream_t stream,
    u32 surfel_count,
    const CUDABuffer<float>& surfels,
    hipGraphicsResource_t neighbor_index_buffer_resource) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  // Map OpenGL buffer object for writing from CUDA.
  hipGraphicsMapResources(1, &neighbor_index_buffer_resource, stream);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  usize num_bytes;
  unsigned int* index_buffer_ptr;
  hipGraphicsResourceGetMappedPointer((void**)&index_buffer_ptr, &num_bytes, neighbor_index_buffer_resource);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  constexpr int kBlockWidth = 1024;
  dim3 grid_dim(GetBlockCount(surfel_count, kBlockWidth));
  dim3 block_dim(kBlockWidth);
  
  UpdateNeighborIndexBufferCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      surfel_count,
      surfels.ToCUDA(),
      index_buffer_ptr);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  hipGraphicsUnmapResources(1, &neighbor_index_buffer_resource, stream);
}


__global__ void UpdateNormalVertexBufferCUDAKernel(
    u32 surfel_count,
    CUDABuffer_<float> surfels,
    float* normal_vertex_buffer_ptr) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    normal_vertex_buffer_ptr[6 * surfel_index + 0] = surfels(kSurfelSmoothX, surfel_index);
    normal_vertex_buffer_ptr[6 * surfel_index + 1] = surfels(kSurfelSmoothY, surfel_index);
    normal_vertex_buffer_ptr[6 * surfel_index + 2] = surfels(kSurfelSmoothZ, surfel_index);
    
    float radius = sqrtf(surfels(kSurfelRadiusSquared, surfel_index));
    normal_vertex_buffer_ptr[6 * surfel_index + 3] = surfels(kSurfelSmoothX, surfel_index) + radius * surfels(kSurfelNormalX, surfel_index);
    normal_vertex_buffer_ptr[6 * surfel_index + 4] = surfels(kSurfelSmoothY, surfel_index) + radius * surfels(kSurfelNormalY, surfel_index);
    normal_vertex_buffer_ptr[6 * surfel_index + 5] = surfels(kSurfelSmoothZ, surfel_index) + radius * surfels(kSurfelNormalZ, surfel_index);
  }
}

void UpdateNormalVertexBufferCUDA(
    hipStream_t stream,
    u32 surfel_count,
    const CUDABuffer<float>& surfels,
    hipGraphicsResource_t normal_vertex_buffer_resource) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  // Map OpenGL buffer object for writing from CUDA.
  hipGraphicsMapResources(1, &normal_vertex_buffer_resource, stream);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  usize num_bytes;
  float* vertex_buffer_ptr;
  hipGraphicsResourceGetMappedPointer((void**)&vertex_buffer_ptr, &num_bytes, normal_vertex_buffer_resource);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  constexpr int kBlockWidth = 1024;
  dim3 grid_dim(GetBlockCount(surfel_count, kBlockWidth));
  dim3 block_dim(kBlockWidth);
  
  UpdateNormalVertexBufferCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      surfel_count,
      surfels.ToCUDA(),
      vertex_buffer_ptr);
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  hipGraphicsUnmapResources(1, &normal_vertex_buffer_resource, stream);
}


__global__ void BlendMeasurementsCUDAStartKernel(
    float depth_scaling,
    CUDABuffer_<u16> depth_buffer,
    CUDABuffer_<u32> supporting_surfels,
    CUDABuffer_<u32> supporting_surfel_counts,
    CUDABuffer_<float> supporting_surfel_depth_sums,
    CUDABuffer_<u8> distance_map,
    CUDABuffer_<float> surfel_depth_average_deltas,
    CUDABuffer_<u8> new_distance_map,
    CUDABuffer_<float> new_surfel_depth_average_deltas) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  constexpr int kBorder = 1;
  if (x >= kBorder && y >= kBorder && x < supporting_surfels.width() - kBorder && y < supporting_surfels.height() - kBorder) {
    // Only consider pixels with valid measurement depth and supporting surfels.
    if (depth_buffer(y, x) == 0 || supporting_surfels(y, x) == Surfel::kInvalidIndex) {
      return;
    }
    
    bool measurement_border_pixel = false;
    bool surfel_border_pixel = false;
    for (int wy = y - 1, wy_end = y + 1; wy <= wy_end; ++ wy) {
      for (int wx = x - 1, wx_end = x + 1; wx <= wx_end; ++ wx) {
        if (depth_buffer(wy, wx) == 0) {
          measurement_border_pixel = true;
        } else if (supporting_surfels(wy, wx) == Surfel::kInvalidIndex) {
          surfel_border_pixel = true;
        }
      }
    }
    
    if (surfel_border_pixel) {
      // TODO: Interpolation should start at the depth after this iteration's integration in this case
      
      new_distance_map(y, x) = 1;
      
      float surfel_depth_average = supporting_surfel_depth_sums(y, x) / supporting_surfel_counts(y, x);
      new_surfel_depth_average_deltas(y, x) = surfel_depth_average - depth_buffer(y, x) / depth_scaling;
    }
    
    if (measurement_border_pixel) {
      distance_map(y, x) = 1;
      
      float surfel_depth_average = supporting_surfel_depth_sums(y, x) / supporting_surfel_counts(y, x);
      surfel_depth_average_deltas(y, x) = surfel_depth_average - depth_buffer(y, x) / depth_scaling;
      
      depth_buffer(y, x) = depth_scaling * surfel_depth_average + 0.5f;  // TODO: This assignment can happen while other threads read, does it matter?
    } else {
      distance_map(y, x) = 255;  // unknown distance
    }
  }
}

__global__ void BlendMeasurementsCUDAIterationKernel(
    int iteration,
    float interpolation_factor_term,
    float depth_scaling,
    CUDABuffer_<u16> depth_buffer,
    CUDABuffer_<u32> supporting_surfels,
    CUDABuffer_<u32> /*supporting_surfel_counts*/,
    CUDABuffer_<float> /*supporting_surfel_depth_sums*/,
    CUDABuffer_<u8> distance_map,
    CUDABuffer_<float> surfel_depth_average_deltas,
    CUDABuffer_<u8> new_distance_map,
    CUDABuffer_<float> new_surfel_depth_average_deltas) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  constexpr int kBorder = 1;
  if (x >= kBorder && y >= kBorder && x < supporting_surfels.width() - kBorder && y < supporting_surfels.height() - kBorder) {
    if (distance_map(y, x) == 255) {  // unknown distance
      float delta_sum = 0;
      int count = 0;
      
      for (int wy = y - 1, wy_end = y + 1; wy <= wy_end; ++ wy) {
        for (int wx = x - 1, wx_end = x + 1; wx <= wx_end; ++ wx) {
          if (distance_map(wy, wx) == iteration - 1) {
            delta_sum += surfel_depth_average_deltas(wy, wx);
            ++ count;
          }
        }
      }
      
      if (count > 0) {
        distance_map(y, x) = iteration;  // TODO: This assignment can happen while other threads read, does it matter?
        float surfel_delta_average = delta_sum / count;
        surfel_depth_average_deltas(y, x) = surfel_delta_average;
        
        float interpolation_factor = (iteration - 1) * interpolation_factor_term;
        depth_buffer(y, x) += depth_scaling * (1 - interpolation_factor) * surfel_delta_average + 0.5f;
      }
    }
    
    if (depth_buffer(y, x) != 0 && supporting_surfels(y, x) == Surfel::kInvalidIndex && new_distance_map(y, x) == 0) {
      float delta_sum = 0;
      int count = 0;
      
      for (int wy = y - 1, wy_end = y + 1; wy <= wy_end; ++ wy) {
        for (int wx = x - 1, wx_end = x + 1; wx <= wx_end; ++ wx) {
          if (new_distance_map(wy, wx) == iteration - 1) {
            delta_sum += new_surfel_depth_average_deltas(wy, wx);
            ++ count;
          }
        }
      }
      
      if (count > 0) {
        new_distance_map(y, x) = iteration;  // TODO: This assignment can happen while other threads read, does it matter?
        float surfel_delta_average = delta_sum / count;
        new_surfel_depth_average_deltas(y, x) = surfel_delta_average;
        
        float interpolation_factor = (iteration - 1) * interpolation_factor_term;
        depth_buffer(y, x) += depth_scaling * (1 - interpolation_factor) * surfel_delta_average + 0.5f;
      }
    }
  }
}

void BlendMeasurementsCUDA(
    hipStream_t stream,
    int measurement_blending_radius,
    float depth_correction_factor,
    CUDABuffer<u16>* depth_buffer,
    const CUDABuffer<u32>& supporting_surfels,
    const CUDABuffer<u32>& supporting_surfel_counts,
    const CUDABuffer<float>& supporting_surfel_depth_sums,
    CUDABuffer<u8>* distance_map,
    CUDABuffer<float>* surfel_depth_average_deltas,
    CUDABuffer<u8>* new_distance_map,
    CUDABuffer<float>* new_surfel_depth_average_deltas) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  distance_map->Clear(0, stream);
  new_distance_map->Clear(0, stream);
  
  constexpr int kBlockWidth = 32;
  constexpr int kBlockHeight = 32;
  dim3 grid_dim(GetBlockCount(supporting_surfels.width(), kBlockWidth),
                GetBlockCount(supporting_surfels.height(), kBlockHeight));
  dim3 block_dim(kBlockWidth, kBlockHeight);
  
  // Find pixels with distance == 1, having a depth measurement next to the measurement border, and supporting surfels.
  BlendMeasurementsCUDAStartKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      1.0f / depth_correction_factor,
      depth_buffer->ToCUDA(),
      supporting_surfels.ToCUDA(),
      supporting_surfel_counts.ToCUDA(),
      supporting_surfel_depth_sums.ToCUDA(),
      distance_map->ToCUDA(),
      surfel_depth_average_deltas->ToCUDA(),
      new_distance_map->ToCUDA(),
      new_surfel_depth_average_deltas->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  // Find pixels with distances in [2, measurement_blending_radius] and average surfel depths.
  for (int iteration = 2; iteration < measurement_blending_radius; ++ iteration) {
    BlendMeasurementsCUDAIterationKernel
    <<<grid_dim, block_dim, 0, stream>>>(
        iteration,
        1.0f / (measurement_blending_radius - 1.0f),
        1.0f / depth_correction_factor,
        depth_buffer->ToCUDA(),
        supporting_surfels.ToCUDA(),
        supporting_surfel_counts.ToCUDA(),
        supporting_surfel_depth_sums.ToCUDA(),
        distance_map->ToCUDA(),
        surfel_depth_average_deltas->ToCUDA(),
        new_distance_map->ToCUDA(),
        new_surfel_depth_average_deltas->ToCUDA());
    #ifdef CUDA_SEQUENTIAL_CHECKS
      hipDeviceSynchronize();
    #endif
    CHECK_CUDA_NO_ERROR();
  }
  
}


__device__ void IntegrateOrConflictSurfel(
    bool integrate, u32 frame_index, int x, int y,
    float fx_inv, float fy_inv, float cx_inv, float cy_inv,
    const float3& cam_space_surfel_pos,
    unsigned int surfel_index,
    CUDABuffer_<float>& surfels,
    const CUDAMatrix3x4& local_T_global,
    const CUDAMatrix3x4& global_T_local,
    float max_surfel_confidence,
    float sensor_noise_factor,
    float cos_normal_compatibility_threshold,
    float depth_correction_factor,
    const CUDABuffer_<u16>& depth_buffer,
    CUDABuffer_<float2> normals_buffer,
    CUDABuffer_<float> radius_buffer,
    CUDABuffer_<uchar3> color_buffer,
    CUDABuffer_<u32>& /*supporting_surfels*/,
    CUDABuffer_<u32>& supporting_surfel_counts,
    CUDABuffer_<u32>& conflicting_surfels,
    CUDABuffer_<float>& first_surfel_depth) {
  // Check whether the surfel falls on a depth pixel.
  float measurement_depth = depth_correction_factor * depth_buffer(y, x);
  if (measurement_depth <= 0) {
    integrate = false;
  }
  if (!__any(integrate)) {
    return;
  }
  
  // Check if this or another surfel is conflicting.
  bool conflicting = false;
  const float first_surfel_depth_value = first_surfel_depth(y, x);
  if (first_surfel_depth_value < (1 - sensor_noise_factor) * measurement_depth) {
    // This or another surfel is conflicting.
    if (first_surfel_depth_value == cam_space_surfel_pos.z) {
      // This surfel is conflicting with the measurement.
      if (conflicting_surfels(y, x) == surfel_index) {
        conflicting = integrate;
      }
    }
    integrate = false;
  }
  if (!__any(integrate || conflicting)) {
    return;
  }
  
  // Determine the depth from which on surfels are considered to be occluded.
  float occlusion_depth = (1 + sensor_noise_factor) * measurement_depth;
  if (kProtectSlightlyOccludedSurfels && first_surfel_depth_value < occlusion_depth) {
    // TODO: Would it be better to use the front surfel's radius for that?
    occlusion_depth = (1 + kOcclusionDepthFactor) * first_surfel_depth_value;
  }
  
  // Check whether this surfel is occluded.
  if (cam_space_surfel_pos.z > occlusion_depth) {
    // Surfel is occluded.
    integrate = false;
  }
  if (!__any(integrate || conflicting)) {
    return;
  }
  
  
  // Read data.
  float depth = depth_correction_factor * depth_buffer(y, x);
  float3 local_position;
  UnprojectPoint(x, y, depth, fx_inv, fy_inv, cx_inv, cy_inv, &local_position);
  float3 global_position = global_T_local * local_position;
  
  float2 normal_xy = normals_buffer(y, x);
  const float normal_z = -sqrtf(::max(0.f, 1 - normal_xy.x * normal_xy.x - normal_xy.y * normal_xy.y));
  float3 global_normal = global_T_local.Rotate(make_float3(normal_xy.x, normal_xy.y, normal_z));
  
  uchar3 color = color_buffer(y, x);
  
  // Handle conflicts.
  // Critical section. HACK: replace surfel x coordinate with NaN to signal locked state.
  __syncthreads();  // Not sure if necessary
  while (__any(conflicting)) {
    float assumed_x = surfels(kSurfelX, surfel_index);
    if (conflicting &&
        !::isnan(assumed_x) &&
        atomicCAS(reinterpret_cast<int*>(&surfels(kSurfelX, surfel_index)),
                  __float_as_int(assumed_x),
                  __float_as_int(HIP_NAN_F)) == __float_as_int(assumed_x)) {
      // Handle the conflict with surfel_index.
      float confidence = surfels(kSurfelConfidence, surfel_index);
      confidence -= 1;
      if (confidence <= 0) {
        // Delete the old surfel by replacing it with a new one.
        assumed_x = global_position.x;
        surfels(kSurfelY, surfel_index) = global_position.y;
        surfels(kSurfelZ, surfel_index) = global_position.z;
        
        surfels(kSurfelSmoothX, surfel_index) = global_position.x;
        surfels(kSurfelSmoothY, surfel_index) = global_position.y;
        surfels(kSurfelSmoothZ, surfel_index) = global_position.z;
        
        surfels(kSurfelNormalX, surfel_index) = global_normal.x;
        surfels(kSurfelNormalY, surfel_index) = global_normal.y;
        surfels(kSurfelNormalZ, surfel_index) = global_normal.z;
        
        *(reinterpret_cast<uchar4*>(&surfels(kSurfelColor, surfel_index))) = make_uchar4(color.x, color.y, color.z, 1);  // Sets the neighbor detach request flag.
        
        surfels(kSurfelRadiusSquared, surfel_index) = radius_buffer(y, x);
        
        #pragma unroll
        for (int i = 0; i < kSurfelNeighborCount; ++ i) {
          // TODO: (Sh/c)ould the neighbors be initialized to something here instead of being removed completely?
          *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + i, surfel_index)) = Surfel::kInvalidIndex;
        }
        
        surfels(kSurfelConfidence, surfel_index) = 1;
        *reinterpret_cast<u32*>(&surfels(kSurfelCreationStamp, surfel_index)) = frame_index;
        *reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, surfel_index)) = frame_index;
      } else {
        surfels(kSurfelConfidence, surfel_index) = confidence;
      }
      
      // Release lock by setting x coordinate.
      // Not sure whether the atomicExch is necessary here, an atomic assignment would suffice.
      atomicExch(reinterpret_cast<int*>(&surfels(kSurfelX, surfel_index)), __float_as_int(assumed_x));
      
      conflicting = false;
    }
    // Force execution of the if case to avoid hang coming from the fact that
    // only the threads which don't go into the if case are executed otherwise.
    __syncthreads();
  }
  
  // Early exit if none of the threads in the warp needs to integrate data.
  if (!__any(integrate)) {
    return;
  }
  
  // The measurement supports the surfel. Determine whether they belong to the
  // same surface (then the measurement should be integrated into the surfel),
  // or to different surfaces (then the measurement must not be integrated).
  
  // Check whether the surfel normal looks towards the camera (instead of away from it).
  float surfel_distance = Norm(cam_space_surfel_pos);
  float3 global_surfel_normal = make_float3(surfels(kSurfelNormalX, surfel_index),
                                            surfels(kSurfelNormalY, surfel_index),
                                            surfels(kSurfelNormalZ, surfel_index));
  float3 local_surfel_normal = local_T_global.Rotate(global_surfel_normal);
  
  float dot_angle = (1.0f / surfel_distance) * (cam_space_surfel_pos.x * local_surfel_normal.x +
                                                cam_space_surfel_pos.y * local_surfel_normal.y +
                                                cam_space_surfel_pos.z * local_surfel_normal.z);
  if (dot_angle > kSurfelNormalToViewingDirThreshold) {
    integrate = false;
  }
  if (!__any(integrate)) {
    return;
  }
  
  // Check whether the surfel normal is compatible with the measurement normal.
  if (measurement_depth < cam_space_surfel_pos.z) {
    float dot_angle = global_surfel_normal.x * global_normal.x +
                      global_surfel_normal.y * global_normal.y +
                      global_surfel_normal.z * global_normal.z;
    if (dot_angle < cos_normal_compatibility_threshold) {
      integrate = false;
    }
  }
  
  // Check whether the observation scale is compatible with the surfel scale.
  const float surfel_radius_squared = surfels(kSurfelRadiusSquared, surfel_index);
  if (surfel_radius_squared < 0) {
    integrate = false;
  }
  if (kCheckScaleCompatibilityForIntegration) {
    const float observation_radius_squared = radius_buffer(y, x);
    if (observation_radius_squared / surfel_radius_squared > kMaxObservationRadiusFactorForIntegration * kMaxObservationRadiusFactorForIntegration) {
      integrate = false;
    }
    if (!__any(integrate)) {
      return;
    }
  }
  
  
  // Integrate.
  // Critical section. HACK: replace surfel x coordinate with NaN to signal locked state.
  __syncthreads();  // Not sure if necessary
  while (__any(integrate)) {
    const float assumed_x = surfels(kSurfelX, surfel_index);
    if (integrate &&
        !::isnan(assumed_x) &&
        atomicCAS(reinterpret_cast<int*>(&surfels(kSurfelX, surfel_index)),
                  __float_as_int(assumed_x),
                  __float_as_int(HIP_NAN_F)) == __float_as_int(assumed_x)) {
      // TODO: Check why this max(1, ...) is necessary
      const float weight = 1.0f / ::max(1, supporting_surfel_counts(y, x));
      
      float new_surfel_x = assumed_x;
      
      // If the surfel has been created (i.e., replaced) in this iteration, do not
      // integrate the data, since the association is probably not valid anymore.
      // Also, the neighbor detach request flag should be kept in that case.
      if (*reinterpret_cast<u32*>(&surfels(kSurfelCreationStamp, surfel_index)) < frame_index) {
        const float confidence = surfels(kSurfelConfidence, surfel_index);
        surfels(kSurfelConfidence, surfel_index) =
            (confidence + weight < max_surfel_confidence) ?
            (confidence + weight) :
            max_surfel_confidence;
        float normalization_factor = 1.0f / (confidence + weight);
        
        new_surfel_x = (confidence * assumed_x + weight * global_position.x) * normalization_factor;  // assumed_x is the old surfel x value.
        surfels(kSurfelY, surfel_index) = (confidence * surfels(kSurfelY, surfel_index) + weight * global_position.y) * normalization_factor;
        surfels(kSurfelZ, surfel_index) = (confidence * surfels(kSurfelZ, surfel_index) + weight * global_position.z) * normalization_factor;
        
        float3 new_normal = make_float3(confidence * surfels(kSurfelNormalX, surfel_index) + weight * global_normal.x,
                                        confidence * surfels(kSurfelNormalY, surfel_index) + weight * global_normal.y,
                                        confidence * surfels(kSurfelNormalZ, surfel_index) + weight * global_normal.z);
        float normal_normalization = 1.0f / sqrtf(new_normal.x * new_normal.x + new_normal.y * new_normal.y + new_normal.z * new_normal.z);
        surfels(kSurfelNormalX, surfel_index) = normal_normalization * new_normal.x;
        surfels(kSurfelNormalY, surfel_index) = normal_normalization * new_normal.y;
        surfels(kSurfelNormalZ, surfel_index) = normal_normalization * new_normal.z;
        
        surfels(kSurfelRadiusSquared, surfel_index) = ::min(surfels(kSurfelRadiusSquared, surfel_index), radius_buffer(y, x));
        
        const uchar4 old_color = *(reinterpret_cast<uchar4*>(&surfels(kSurfelColor, surfel_index)));
        const uchar3 new_color = make_uchar3(
            (confidence * old_color.x + weight * color.x) * normalization_factor + 0.5f,
            (confidence * old_color.y + weight * color.y) * normalization_factor + 0.5f,
            (confidence * old_color.z + weight * color.z) * normalization_factor + 0.5f);
        *(reinterpret_cast<uchar4*>(&surfels(kSurfelColor, surfel_index))) = make_uchar4(new_color.x, new_color.y, new_color.z, 0);  // NOTE: Unsets the neighbor detach request flag
        
        *reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, surfel_index)) = frame_index;
      }
      
      // Release lock by setting x coordinate.
      // Not sure whether the atomicExch is necessary here, an atomic assignment would suffice.
      atomicExch(reinterpret_cast<int*>(&surfels(kSurfelX, surfel_index)), __float_as_int(new_surfel_x));
      
      integrate = false;
    }
    // Force execution of the if case to avoid hang coming from the fact that
    // only the threads which don't go into the if case are executed otherwise.
    __syncthreads();
  }
}

__global__ void IntegrateMeasurementsCUDAKernel(
    u32 frame_index,
    int surfel_integration_active_window_size,
    float max_surfel_confidence,
    float sensor_noise_factor,
    float cos_normal_compatibility_threshold,
    float inv_depth_scaling,
    float fx, float fy, float cx, float cy,
    float fx_inv, float fy_inv, float cx_inv, float cy_inv,
    CUDAMatrix3x4 local_T_global,
    CUDAMatrix3x4 global_T_local,
    CUDABuffer_<u16> depth_buffer,
    CUDABuffer_<float2> normals_buffer,
    CUDABuffer_<float> radius_buffer,
    CUDABuffer_<uchar3> color_buffer,
    CUDABuffer_<u32> supporting_surfels,
    CUDABuffer_<u32> supporting_surfel_counts,
    CUDABuffer_<u32> conflicting_surfels,
    CUDABuffer_<float> first_surfel_depth,
    u32 surfel_count,
    CUDABuffer_<float> surfels) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  bool integrate = true;
  
  // Check whether the surfel projects onto the image. Keep all threads active
  // such that the __syncthreads() later will work.
  if (surfel_index >= surfel_count) {
    surfel_index = 0;
    integrate = false;
  }
  
  if (!IsSurfelActiveForIntegration(surfel_index, surfels, frame_index, surfel_integration_active_window_size)) {
    integrate = false;
  }
  if (!__any(integrate)) {
    return;
  }
  
  float3 global_position =
      make_float3(surfels(kSurfelX, surfel_index),
                  surfels(kSurfelY, surfel_index),
                  surfels(kSurfelZ, surfel_index));
  float3 local_position = local_T_global * global_position;
  if (local_position.z <= 0) {
    // TODO: Compute z before x and y such that this early exit decision can be done earlier?
    integrate = false;
  }
  // Early exit?
  if (!__any(integrate)) {
    return;
  }
  
  float2 pixel_pos =
      make_float2(fx * (local_position.x / local_position.z) + cx,
                  fy * (local_position.y / local_position.z) + cy);
  int px = static_cast<int>(pixel_pos.x);
  int py = static_cast<int>(pixel_pos.y);
  if (pixel_pos.x < 0 || pixel_pos.y < 0 ||
      px < 0 || py < 0 ||
      px >= depth_buffer.width() || py >= depth_buffer.height()) {
    px = 0;
    py = 0;
    integrate = false;
  }
  
  if (surfels(kSurfelRadiusSquared, surfel_index) < 0) {
    integrate = false;
  }
  
  // Early exit?
  if (!__any(integrate)) {
    return;
  }
  
  IntegrateOrConflictSurfel(
      integrate, frame_index, px, py,
      fx_inv, fy_inv, cx_inv, cy_inv,
      local_position,
      surfel_index, surfels,
      local_T_global,
      global_T_local,
      max_surfel_confidence,
      sensor_noise_factor,
      cos_normal_compatibility_threshold,
      inv_depth_scaling, depth_buffer,
      normals_buffer,
      radius_buffer,
      color_buffer,
      supporting_surfels,
      supporting_surfel_counts,
      conflicting_surfels,
      first_surfel_depth);
  
  float x_frac = pixel_pos.x - px;
  float y_frac = pixel_pos.y - py;
  int offset_x = 0;
  int offset_y = 0;
  if (x_frac < y_frac) {
    // Surfel is within the bottom-left triangle half of the pixel.
    if (x_frac < 1 - y_frac) {
      // Surfel is on the left side of the pixel.
      if (px > 1) {
        offset_x = px - 1;
        offset_y = py;
      } else {
        integrate = false;
      }
    } else {
      // Surfel is on the bottom side of the pixel.
      if (py < depth_buffer.height() - 1) {
        offset_x = px;
        offset_y = py + 1;
      } else {
        integrate = false;
      }
    }
  } else {
    // Surfel is within the top-right triangle half of the pixel.
    if (x_frac < 1 - y_frac) {
      // Surfel is on the top side of the pixel.
      if (py > 0) {
        offset_x = px;
        offset_y = py - 1;
      } else {
        integrate = false;
      }
    } else {
      // Surfel is on the right side of the pixel.
      if (px < depth_buffer.width() - 1) {
        offset_x = px + 1;
        offset_y = py;
      } else {
        integrate = false;
      }
    }
  }
  
  IntegrateOrConflictSurfel(
      integrate, frame_index, offset_x, offset_y,
      fx_inv, fy_inv, cx_inv, cy_inv,
      local_position,
      surfel_index, surfels,
      local_T_global,
      global_T_local,
      max_surfel_confidence,
      sensor_noise_factor,
      cos_normal_compatibility_threshold,
      inv_depth_scaling, depth_buffer,
      normals_buffer,
      radius_buffer,
      color_buffer,
      supporting_surfels,
      supporting_surfel_counts,
      conflicting_surfels,
      first_surfel_depth);
  
  // TODO: use half integration weight if the surfel is associated to two pixels?
}

void IntegrateMeasurementsCUDA(
    hipStream_t stream,
    u32 frame_index,
    int surfel_integration_active_window_size,
    float max_surfel_confidence,
    float sensor_noise_factor,
    float normal_compatibility_threshold_deg,
    const SE3f& global_T_local,
    float depth_scaling,
    const PinholeCamera4f& depth_camera,
    const CUDABuffer<u16>& depth_buffer,
    const CUDABuffer<float2>& normals_buffer,
    const CUDABuffer<float>& radius_buffer,
    const CUDABuffer<Vec3u8>& color_buffer,
    const CUDABuffer<u32>& supporting_surfels,
    const CUDABuffer<u32>& supporting_surfel_counts,
    const CUDABuffer<u32>& conflicting_surfels,
    const CUDABuffer<float>& first_surfel_depth,
    u32 surfel_count,
    CUDABuffer<float>* surfels) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  const float fx = depth_camera.parameters()[0];
  const float fy = depth_camera.parameters()[1];
  const float cx = depth_camera.parameters()[2];
  const float cy = depth_camera.parameters()[3];
  
  // Unprojection intrinsics for pixel center convention.
  const float fx_inv = 1.0f / fx;
  const float fy_inv = 1.0f / fy;
  const float cx_pixel_center = cx - 0.5f;
  const float cy_pixel_center = cy - 0.5f;
  const float cx_inv_pixel_center = -cx_pixel_center / fx;
  const float cy_inv_pixel_center = -cy_pixel_center / fy;
  
  //constexpr int kBlockWidth = 1024;
  constexpr int kBlockWidth = 32;
  dim3 grid_dim(GetBlockCount(surfel_count, kBlockWidth));
  dim3 block_dim(kBlockWidth);
  
  IntegrateMeasurementsCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      surfel_integration_active_window_size,
      max_surfel_confidence,
      sensor_noise_factor,
      cosf(M_PI / 180.0f * normal_compatibility_threshold_deg),
      1.0f / depth_scaling,
      fx, fy, cx, cy,
      fx_inv, fy_inv, cx_inv_pixel_center, cy_inv_pixel_center,
      CUDAMatrix3x4(global_T_local.inverse().matrix3x4()),
      CUDAMatrix3x4(global_T_local.matrix3x4()),
      depth_buffer.ToCUDA(),
      normals_buffer.ToCUDA(),
      radius_buffer.ToCUDA(),
      *reinterpret_cast<const CUDABuffer_<uchar3>*>(&color_buffer.ToCUDA()),
      supporting_surfels.ToCUDA(),
      supporting_surfel_counts.ToCUDA(),
      conflicting_surfels.ToCUDA(),
      first_surfel_depth.ToCUDA(),
      surfel_count,
      surfels->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
}


__global__ void UpdateNeighborsCUDAKernel(
    u32 frame_index,
    int surfel_integration_active_window_size,
    float radius_factor_for_regularization_neighbors_squared,
    CUDABuffer_<u32> supporting_surfels,
    CUDABuffer_<u32> /*conflicting_surfels*/,
    float fx, float fy, float cx, float cy,
    CUDAMatrix3x4 local_T_global,
    float sensor_noise_factor,
    float depth_correction_factor,
    CUDABuffer_<u16> depth_buffer,
    CUDABuffer_<float2> /*normals_buffer*/,
    CUDABuffer_<float> radius_buffer,
    CUDABuffer_<float> first_surfel_depth,
    u32 surfel_count,
    CUDABuffer_<float> surfels) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    if (!IsSurfelActiveForIntegration(surfel_index, surfels, frame_index, surfel_integration_active_window_size)) {
      return;
    }
    
    // Project the surfel into the image.
    float3 global_position =
        make_float3(surfels(kSurfelX, surfel_index),
                    surfels(kSurfelY, surfel_index),
                    surfels(kSurfelZ, surfel_index));
    float3 cam_space_surfel_pos = local_T_global * global_position;
    if (cam_space_surfel_pos.z <= 0) {
      // TODO: Compute z before x and y such that this early exit decision can be done earlier?
      return;
    }
    
    float2 pixel_pos =
        make_float2(fx * (cam_space_surfel_pos.x / cam_space_surfel_pos.z) + cx,
                    fy * (cam_space_surfel_pos.y / cam_space_surfel_pos.z) + cy);
    int x = static_cast<int>(pixel_pos.x);
    int y = static_cast<int>(pixel_pos.y);
    
    // Use 1 pixel border.
    constexpr int kBorder = 1;
    if (x < kBorder || y < kBorder ||
        x >= supporting_surfels.width() - kBorder || y >= supporting_surfels.height() - kBorder) {
      return;
    }
    
    // Is the surfel occluded?
    float measurement_depth = depth_correction_factor * depth_buffer(y, x);
    float occlusion_depth = (1 + sensor_noise_factor) * measurement_depth;
    if (kProtectSlightlyOccludedSurfels) {
      const float first_surfel_depth_value = first_surfel_depth(y, x);
      if (first_surfel_depth_value < occlusion_depth) {
        // TODO: Would it be better to use the front surfel's radius for that?
        occlusion_depth = (1 + kOcclusionDepthFactor) * first_surfel_depth_value;
      }
    }
    if (cam_space_surfel_pos.z > occlusion_depth) {
      return;
    }
    
    // Check whether the surfel normal looks towards the camera (instead of away from it).
    float surfel_distance = Norm(cam_space_surfel_pos);
    float3 global_surfel_normal = make_float3(surfels(kSurfelNormalX, surfel_index),
                                              surfels(kSurfelNormalY, surfel_index),
                                              surfels(kSurfelNormalZ, surfel_index));
    float3 local_surfel_normal = local_T_global.Rotate(global_surfel_normal);
    
    float dot_angle = (1.0f / surfel_distance) * (cam_space_surfel_pos.x * local_surfel_normal.x +
                                                  cam_space_surfel_pos.y * local_surfel_normal.y +
                                                  cam_space_surfel_pos.z * local_surfel_normal.z);
    if (dot_angle > kSurfelNormalToViewingDirThreshold) {
      return;
    }
    
    // Check whether the surfel normal is compatible with the measurement normal (if enabled).
    /*if (measurement_depth < cam_space_surfel_pos.z) {
      float2 normal = normals_buffer(y, x);
      float3 local_normal = make_float3(normal.x, normal.y, -sqrtf(::max(0.f, 1 - normal.x * normal.x - normal.y * normal.y)));
      
      float dot_angle = local_surfel_normal.x * local_normal.x +
                        local_surfel_normal.y * local_normal.y +
                        local_surfel_normal.z * local_normal.z;
      if (dot_angle < kNormalCompatibilityThreshold) {
        return;
      }
    }*/
    
    const float surfel_radius_squared = surfels(kSurfelRadiusSquared, surfel_index);
    if (surfel_radius_squared < 0) {
      return;
    }
    if (kCheckScaleCompatibilityForNeighborAssignment) {
      const float observation_radius_squared = radius_buffer(y, x);
      if (observation_radius_squared / surfel_radius_squared > kMaxObservationRadiusFactorForIntegration * kMaxObservationRadiusFactorForIntegration) {
        return;
      }
    }
    
    // We think that the surfel is visible, update its neighbors.
    
    float radius_squared = surfels(kSurfelRadiusSquared, surfel_index);
    
    float3 global_normal =
        make_float3(surfels(kSurfelNormalX, surfel_index),
                    surfels(kSurfelNormalY, surfel_index),
                    surfels(kSurfelNormalZ, surfel_index));
    
    // Compute distances to existing neighbors.
    float neighbor_distances_squared[kSurfelNeighborCount];
    u32 neighbor_surfel_indices[kSurfelNeighborCount];
    for (int n = 0; n < kSurfelNeighborCount; ++ n) {
      neighbor_surfel_indices[n] = *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + n, surfel_index));
      if (neighbor_surfel_indices[n] == Surfel::kInvalidIndex) {
        neighbor_distances_squared[n] = HIP_INF_F;
      } else {
        float3 neighbor_position =
            make_float3(surfels(kSurfelX, neighbor_surfel_indices[n]),
                        surfels(kSurfelY, neighbor_surfel_indices[n]),
                        surfels(kSurfelZ, neighbor_surfel_indices[n]));
        float3 surfel_to_neighbor = make_float3(
            global_position.x - neighbor_position.x,
            global_position.y - neighbor_position.y,
            global_position.z - neighbor_position.z);
        neighbor_distances_squared[n] = surfel_to_neighbor.x * surfel_to_neighbor.x + surfel_to_neighbor.y * surfel_to_neighbor.y + surfel_to_neighbor.z * surfel_to_neighbor.z;
      }
    }
    
    constexpr int kDirectionsX[4] = {-1, 1, 0, 0};
    constexpr int kDirectionsY[4] = {0, 0, -1, 1};
    for (int direction = 0; direction < 4; ++ direction) {
      u32 neighbor_index = supporting_surfels(y + kDirectionsY[direction], x + kDirectionsX[direction]);
      if (neighbor_index != Surfel::kInvalidIndex &&
          neighbor_index != surfel_index) {
        // Check for closeness.
        float3 this_to_neighbor = make_float3(surfels(kSurfelX, neighbor_index) - global_position.x,
                                              surfels(kSurfelY, neighbor_index) - global_position.y,
                                              surfels(kSurfelZ, neighbor_index) - global_position.z);
        float distance_squared =
            this_to_neighbor.x * this_to_neighbor.x + this_to_neighbor.y * this_to_neighbor.y + this_to_neighbor.z * this_to_neighbor.z;
        if (distance_squared > radius_factor_for_regularization_neighbors_squared * radius_squared) {
          neighbor_index = Surfel::kInvalidIndex;
        }
        
        if (neighbor_index != Surfel::kInvalidIndex) {
          // Check for compatible normal.
          float3 neighbor_normal =
              make_float3(surfels(kSurfelNormalX, neighbor_index),
                          surfels(kSurfelNormalY, neighbor_index),
                          surfels(kSurfelNormalZ, neighbor_index));
          float normal_dot = global_normal.x * neighbor_normal.x +
                             global_normal.y * neighbor_normal.y +
                             global_normal.z * neighbor_normal.z;
          if (normal_dot <= 0) {
            neighbor_index = Surfel::kInvalidIndex;
          }
          
          if (neighbor_index != Surfel::kInvalidIndex) {
            // Check whether it is already a neighbor, or find the best insertion slot.
            int best_n = -1;
            float best_distance_squared = -1;
            for (int n = 0; n < kSurfelNeighborCount; ++ n) {
              if (neighbor_index == neighbor_surfel_indices[n]) {
                best_n = -1;
                break;
              } else if (neighbor_distances_squared[n] > best_distance_squared) {
                best_n = n;
                best_distance_squared = neighbor_distances_squared[n];
              }
            }
            
            if (best_n >= 0 && distance_squared < best_distance_squared) {
              neighbor_surfel_indices[best_n] = neighbor_index;
              neighbor_distances_squared[best_n] = distance_squared;
            }
          }
        }
      }
    }
    
    // Write the neighbor indices back to global memory.
    for (int n = 0; n < kSurfelNeighborCount; ++ n) {
      *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + n, surfel_index)) = neighbor_surfel_indices[n];
    }
  }
}

__global__ void UpdateNeighborsCUDARemoveReplacedNeighborsKernel(
    u32 frame_index,
    u32 surfel_count,
    CUDABuffer_<float> surfels) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    for (int neighbor_index = 0; neighbor_index < kSurfelNeighborCount; ++ neighbor_index) {
      u32 neighbor_surfel_index = *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + neighbor_index, surfel_index));
      if (neighbor_surfel_index != Surfel::kInvalidIndex) {
        if (*reinterpret_cast<u8*>(&reinterpret_cast<uchar4*>(&surfels(kSurfelColor, neighbor_surfel_index))->w) == 1) {
          // This neighbor has the neighbor detach request flag set. Remove it.
          *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + neighbor_index, surfel_index)) = Surfel::kInvalidIndex;
        }
      }
    }
  }
}

void UpdateNeighborsCUDA(
    hipStream_t stream,
    u32 frame_index,
    int surfel_integration_active_window_size,
    float radius_factor_for_regularization_neighbors,
    const CUDABuffer<u32>& supporting_surfels,
    const CUDABuffer<u32>& conflicting_surfels,
    const PinholeCamera4f& depth_camera,
    const SE3f& local_T_global,
    float sensor_noise_factor,
    float depth_correction_factor,
    const CUDABuffer<u16>& depth_buffer,
    const CUDABuffer<float2>& normals_buffer,
    const CUDABuffer<float>& radius_buffer,
    const CUDABuffer<float>& first_surfel_depth,
    usize surfel_count,
    CUDABuffer<float>* surfels) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  const float fx = depth_camera.parameters()[0];
  const float fy = depth_camera.parameters()[1];
  const float cx = depth_camera.parameters()[2];
  const float cy = depth_camera.parameters()[3];
  
  constexpr int kSurfelsBlockWidth = 1024;
  dim3 grid_dim(GetBlockCount(surfel_count, kSurfelsBlockWidth));
  dim3 block_dim(kSurfelsBlockWidth);
  
  UpdateNeighborsCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      surfel_integration_active_window_size,
      radius_factor_for_regularization_neighbors * radius_factor_for_regularization_neighbors,
      supporting_surfels.ToCUDA(),
      conflicting_surfels.ToCUDA(),
      fx, fy, cx, cy,
      CUDAMatrix3x4(local_T_global.matrix3x4()),
      sensor_noise_factor,
      depth_correction_factor,
      depth_buffer.ToCUDA(),
      normals_buffer.ToCUDA(),
      radius_buffer.ToCUDA(),
      first_surfel_depth.ToCUDA(),
      surfel_count,
      surfels->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  
  UpdateNeighborsCUDARemoveReplacedNeighborsKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      surfel_count,
      surfels->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
}


__forceinline__ __device__ void RenderMinDepthAtPixel(
    int x, int y,
    const float3& cam_space_surfel_pos,
    CUDABuffer_<float>& first_surfel_depth) {
  // Should behave properly as long as all the floats are positive.
  atomicMin(reinterpret_cast<int*>(&first_surfel_depth(y, x)), __float_as_int(cam_space_surfel_pos.z));
}

__global__ void RenderMinDepthCUDAKernel(
    u32 frame_index,
    int surfel_integration_active_window_size,
    float fx, float fy, float cx, float cy,
    CUDAMatrix3x4 local_T_global,
    u32 surfel_count,
    CUDABuffer_<float> surfels,
    CUDABuffer_<float> first_surfel_depth) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    if (!IsSurfelActiveForIntegration(surfel_index, surfels, frame_index, surfel_integration_active_window_size)) {
      return;
    }
    
    float3 global_position =
        make_float3(surfels(kSurfelX, surfel_index),
                    surfels(kSurfelY, surfel_index),
                    surfels(kSurfelZ, surfel_index));
    float3 local_position = local_T_global * global_position;
    if (local_position.z <= 0) {
      // TODO: Compute z before x and y such that this early exit can be done earlier?
      return;
    }
    
    float2 pixel_pos =
        make_float2(fx * (local_position.x / local_position.z) + cx,
                    fy * (local_position.y / local_position.z) + cy);
    int px = static_cast<int>(pixel_pos.x);
    int py = static_cast<int>(pixel_pos.y);
    if (pixel_pos.x < 0 || pixel_pos.y < 0 ||
        px < 0 || py < 0 ||
        px >= first_surfel_depth.width() || py >= first_surfel_depth.height()) {
      return;
    }
    
    RenderMinDepthAtPixel(
        px, py, local_position,
        first_surfel_depth);
    
    float x_frac = pixel_pos.x - px;
    float y_frac = pixel_pos.y - py;
    bool integrate = true;
    int offset_x;
    int offset_y;
    if (x_frac < y_frac) {
      // Surfel is within the bottom-left triangle half of the pixel.
      if (x_frac < 1 - y_frac) {
        // Surfel is on the left side of the pixel.
        if (px > 1) {
          offset_x = px - 1;
          offset_y = py;
        } else {
          integrate = false;
        }
      } else {
        // Surfel is on the bottom side of the pixel.
        if (py < first_surfel_depth.height() - 1) {
          offset_x = px;
          offset_y = py + 1;
        } else {
          integrate = false;
        }
      }
    } else {
      // Surfel is within the top-right triangle half of the pixel.
      if (x_frac < 1 - y_frac) {
        // Surfel is on the top side of the pixel.
        if (py > 0) {
          offset_x = px;
          offset_y = py - 1;
        } else {
          integrate = false;
        }
      } else {
        // Surfel is on the right side of the pixel.
        if (px < first_surfel_depth.width() - 1) {
          offset_x = px + 1;
          offset_y = py;
        } else {
          integrate = false;
        }
      }
    }
    
    if (integrate) {
      RenderMinDepthAtPixel(
          offset_x, offset_y, local_position,
          first_surfel_depth);
    }
  }
}

void RenderMinDepthCUDA(
    hipStream_t stream,
    u32 frame_index,
    int surfel_integration_active_window_size,
    const SE3f& local_T_global,
    const PinholeCamera4f& depth_camera,
    CUDABuffer<float>* first_surfel_depth,
    u32 surfel_count,
    const CUDABuffer<float>& surfels) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  const float fx = depth_camera.parameters()[0];
  const float fy = depth_camera.parameters()[1];
  const float cx = depth_camera.parameters()[2];
  const float cy = depth_camera.parameters()[3];
  
  constexpr int kBlockWidth = 1024;
  dim3 grid_dim(GetBlockCount(surfel_count, kBlockWidth));
  dim3 block_dim(kBlockWidth);
  
  RenderMinDepthCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      surfel_integration_active_window_size,
      fx, fy, cx, cy,
      CUDAMatrix3x4(local_T_global.matrix3x4()),
      surfel_count,
      surfels.ToCUDA(),
      first_surfel_depth->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
}


__device__ void ConsiderSurfelAssociationToPixel(
    int x, int y,
    const float3& cam_space_surfel_pos,
    unsigned int surfel_index,
    const CUDABuffer_<float>& surfels,
    const CUDAMatrix3x4& local_T_global,
    float sensor_noise_factor,
    float cos_normal_compatibility_threshold,
    float depth_correction_factor,
    const CUDABuffer_<u16>& depth_buffer,
    const CUDABuffer_<float2>& normals_buffer,
    const CUDABuffer_<float>& radius_buffer,
    CUDABuffer_<u32>& supporting_surfels,
    CUDABuffer_<u32>& supporting_surfel_counts,
    CUDABuffer_<float>& supporting_surfel_depth_sums,
    CUDABuffer_<u32>& conflicting_surfels,
    CUDABuffer_<float>& first_surfel_depth) {
  // Check whether the surfel falls on a depth pixel.
  float measurement_depth = depth_correction_factor * depth_buffer(y, x);
  if (measurement_depth <= 0) {
    return;
  }
  
  // Check if this or another surfel is conflicting.
  const float first_surfel_depth_value = first_surfel_depth(y, x);
  if (first_surfel_depth_value < (1 - sensor_noise_factor) * measurement_depth) {
    // This or another surfel is conflicting.
    if (first_surfel_depth_value == cam_space_surfel_pos.z) {
      // This surfel is conflicting.
      conflicting_surfels(y, x) = surfel_index;
    }
    return;
  }
  
  // Determine the depth from which on surfels are considered to be occluded.
  float occlusion_depth = (1 + sensor_noise_factor) * measurement_depth;
  if (kProtectSlightlyOccludedSurfels) {
    if (first_surfel_depth_value < occlusion_depth) {
      // TODO: Would it be better to use the front surfel's radius for that?
      occlusion_depth = (1 + kOcclusionDepthFactor) * first_surfel_depth_value;
    }
  }
  
  // Check if this surfel is occluded.
  if (cam_space_surfel_pos.z > occlusion_depth) {
    // Surfel is occluded.
    return;
  }
  
  // The measurement supports the surfel. Determine whether they belong to the
  // same surface (then the measurement should be integrated into the surfel),
  // or to different surfaces (then the measurement must not be integrated).
  
  // Check whether the surfel normal looks towards the camera (instead of away from it).
  float surfel_distance = Norm(cam_space_surfel_pos);
  float3 global_surfel_normal = make_float3(surfels(kSurfelNormalX, surfel_index),
                                            surfels(kSurfelNormalY, surfel_index),
                                            surfels(kSurfelNormalZ, surfel_index));
  float3 local_surfel_normal = local_T_global.Rotate(global_surfel_normal);
  
  float dot_angle = (1.0f / surfel_distance) * (cam_space_surfel_pos.x * local_surfel_normal.x +
                                                cam_space_surfel_pos.y * local_surfel_normal.y +
                                                cam_space_surfel_pos.z * local_surfel_normal.z);
  if (dot_angle > kSurfelNormalToViewingDirThreshold) {
    return;
  }
  
  // Check whether the surfel normal is compatible with the measurement normal.
  if (measurement_depth < cam_space_surfel_pos.z) {
    float2 normal = normals_buffer(y, x);
    float3 local_normal = make_float3(normal.x, normal.y, -sqrtf(::max(0.f, 1 - normal.x * normal.x - normal.y * normal.y)));
    
    float dot_angle = local_surfel_normal.x * local_normal.x +
                      local_surfel_normal.y * local_normal.y +
                      local_surfel_normal.z * local_normal.z;
    if (dot_angle < cos_normal_compatibility_threshold) {
      // HACK: Avoid creation of a new surfel here in case there is no other conflicting or supporting surfel
      //       by setting conflicting_surfels(y, x) to an invalid index unequal to Surfel::kInvalidIndex.
      // TODO: This can be harmful since it can prevent the creation of valid surfaces. Delete it?
//       atomicCAS(&conflicting_surfels(y, x), Surfel::kInvalidIndex, Surfel::kInvalidIndex - 1);
      return;
    }
  }
  
  // The measurement seems to belong to the same surface as the surfel.
  
  // Check whether the observation scale is compatible with the surfel scale.
  const float surfel_radius_squared = surfels(kSurfelRadiusSquared, surfel_index);
  if (surfel_radius_squared <= 0) {
    return;
  }
  if (kCheckScaleCompatibilityForIntegration) {
    const float observation_radius_squared = radius_buffer(y, x);
    if (observation_radius_squared / surfel_radius_squared > kMaxObservationRadiusFactorForIntegration * kMaxObservationRadiusFactorForIntegration) {
      // HACK: Avoid creation of a new surfel here in case there is no other conflicting or supporting surfel
      //       by setting conflicting_surfels(y, x) to an invalid index unequal to Surfel::kInvalidIndex.
      atomicCAS(&conflicting_surfels(y, x), Surfel::kInvalidIndex, Surfel::kInvalidIndex - 1);
      return;
    }
  }
  
  // Replace the supporting surfel entry only if it was previously empty
  atomicCAS(&supporting_surfels(y, x), Surfel::kInvalidIndex, surfel_index);
  
  // Add to supporting surfel count for the pixel
  atomicAdd(&supporting_surfel_counts(y, x), 1);
  
  // Add to the supporting surfel depth sum for the pixel
  atomicAdd(&supporting_surfel_depth_sums(y, x), cam_space_surfel_pos.z);
}

__global__ void AssociateSurfelsCUDAKernel(
    u32 frame_index,
    int surfel_integration_active_window_size,
    float fx, float fy, float cx, float cy,
    CUDAMatrix3x4 local_T_global,
    float sensor_noise_factor,
    float cos_normal_compatibility_threshold,
    u32 surfel_count,
    CUDABuffer_<float> surfels,
    float depth_correction_factor,
    CUDABuffer_<u16> depth_buffer,
    CUDABuffer_<float2> normals_buffer,
    CUDABuffer_<float> radius_buffer,
    CUDABuffer_<u32> supporting_surfels,
    CUDABuffer_<u32> supporting_surfel_counts,
    CUDABuffer_<float> supporting_surfel_depth_sums,
    CUDABuffer_<u32> conflicting_surfels,
    CUDABuffer_<float> first_surfel_depth) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    if (!IsSurfelActiveForIntegration(surfel_index, surfels, frame_index, surfel_integration_active_window_size)) {
      return;
    }
    
    float3 global_position =
        make_float3(surfels(kSurfelX, surfel_index),
                    surfels(kSurfelY, surfel_index),
                    surfels(kSurfelZ, surfel_index));
    float3 local_position = local_T_global * global_position;
    if (local_position.z <= 0) {
      // TODO: Compute z before x and y such that this early exit can be done earlier?
      return;
    }
    
    float2 pixel_pos =
        make_float2(fx * (local_position.x / local_position.z) + cx,
                    fy * (local_position.y / local_position.z) + cy);
    int px = static_cast<int>(pixel_pos.x);
    int py = static_cast<int>(pixel_pos.y);
    if (pixel_pos.x < 0 || pixel_pos.y < 0 ||
        px < 0 || py < 0 ||
        px >= depth_buffer.width() || py >= depth_buffer.height()) {
      return;
    }
    
    ConsiderSurfelAssociationToPixel(
        px, py, local_position,
        surfel_index, surfels,
        local_T_global,
        sensor_noise_factor,
        cos_normal_compatibility_threshold,
        depth_correction_factor, depth_buffer, normals_buffer, radius_buffer, supporting_surfels,
        supporting_surfel_counts, supporting_surfel_depth_sums, conflicting_surfels, first_surfel_depth);
    
    float x_frac = pixel_pos.x - px;
    float y_frac = pixel_pos.y - py;
    bool integrate = true;
    int offset_x;
    int offset_y;
    if (x_frac < y_frac) {
      // Surfel is within the bottom-left triangle half of the pixel.
      if (x_frac < 1 - y_frac) {
        // Surfel is on the left side of the pixel.
        if (px > 1) {
          offset_x = px - 1;
          offset_y = py;
        } else {
          integrate = false;
        }
      } else {
        // Surfel is on the bottom side of the pixel.
        if (py < depth_buffer.height() - 1) {
          offset_x = px;
          offset_y = py + 1;
        } else {
          integrate = false;
        }
      }
    } else {
      // Surfel is within the top-right triangle half of the pixel.
      if (x_frac < 1 - y_frac) {
        // Surfel is on the top side of the pixel.
        if (py > 0) {
          offset_x = px;
          offset_y = py - 1;
        } else {
          integrate = false;
        }
      } else {
        // Surfel is on the right side of the pixel.
        if (px < depth_buffer.width() - 1) {
          offset_x = px + 1;
          offset_y = py;
        } else {
          integrate = false;
        }
      }
    }
    
    if (integrate) {
      ConsiderSurfelAssociationToPixel(
          offset_x, offset_y, local_position,
          surfel_index, surfels,
          local_T_global,
          sensor_noise_factor,
          cos_normal_compatibility_threshold,
          depth_correction_factor, depth_buffer, normals_buffer, radius_buffer, supporting_surfels,
          supporting_surfel_counts, supporting_surfel_depth_sums, conflicting_surfels, first_surfel_depth);
    }
  }
}

void AssociateSurfelsCUDA(
    hipStream_t stream,
    u32 frame_index,
    int surfel_integration_active_window_size,
    float sensor_noise_factor,
    float normal_compatibility_threshold_deg,
    const SE3f& local_T_global,
    const PinholeCamera4f& depth_camera,
    float depth_correction_factor,
    const CUDABuffer<u16>& depth_buffer,
    const CUDABuffer<float2>& normals_buffer,
    const CUDABuffer<float>& radius_buffer,
    CUDABuffer<u32>* supporting_surfels,
    CUDABuffer<u32>* supporting_surfel_counts,
    CUDABuffer<float>* supporting_surfel_depth_sums,
    CUDABuffer<u32>* conflicting_surfels,
    CUDABuffer<float>* first_surfel_depth,
    u32 surfel_count,
    const CUDABuffer<float>& surfels) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  const float fx = depth_camera.parameters()[0];
  const float fy = depth_camera.parameters()[1];
  const float cx = depth_camera.parameters()[2];
  const float cy = depth_camera.parameters()[3];
  
  constexpr int kBlockWidth = 1024;
  dim3 grid_dim(GetBlockCount(surfel_count, kBlockWidth));
  dim3 block_dim(kBlockWidth);
  
  AssociateSurfelsCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      surfel_integration_active_window_size,
      fx, fy, cx, cy,
      CUDAMatrix3x4(local_T_global.matrix3x4()),
      sensor_noise_factor,
      cosf(M_PI / 180.0f * normal_compatibility_threshold_deg),
      surfel_count,
      surfels.ToCUDA(),
      depth_correction_factor,
      depth_buffer.ToCUDA(),
      normals_buffer.ToCUDA(),
      radius_buffer.ToCUDA(),
      supporting_surfels->ToCUDA(),
      supporting_surfel_counts->ToCUDA(),
      supporting_surfel_depth_sums->ToCUDA(),
      conflicting_surfels->ToCUDA(),
      first_surfel_depth->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
}


constexpr int kMergeBlockWidth = 1024;

__device__ bool ConsiderSurfelMergeAtPixel(
    int x, int y,
    const float3& cam_space_surfel_pos,
    const float3& global_surfel_pos,
    unsigned int surfel_index,
    CUDABuffer_<float>& surfels,
    const CUDAMatrix3x4& local_T_global,
    float sensor_noise_factor,
    float cos_normal_compatibility_threshold,
    float depth_correction_factor,
    const CUDABuffer_<u16>& depth_buffer,
    const CUDABuffer_<float2>& normals_buffer,
    const CUDABuffer_<float>& radius_buffer,
    CUDABuffer_<u32>& supporting_surfels,
    CUDABuffer_<u32>& supporting_surfel_counts,
    CUDABuffer_<float>& supporting_surfel_depth_sums,
    CUDABuffer_<u32>& conflicting_surfels,
    CUDABuffer_<float>& first_surfel_depth) {
  // Check whether the surfel falls on a depth pixel.
  float measurement_depth = depth_correction_factor * depth_buffer(y, x);
  if (measurement_depth <= 0) {
    return false;
  }
  
  // Check if this or another surfel is conflicting.
  const float first_surfel_depth_value = first_surfel_depth(y, x);
  if (first_surfel_depth_value < (1 - sensor_noise_factor) * measurement_depth) {
    // This or another surfel is conflicting.
    if (first_surfel_depth_value == cam_space_surfel_pos.z) {
      // This surfel is conflicting.
      conflicting_surfels(y, x) = surfel_index;
    }
    return false;
  }
  
  // Determine the depth from which on surfels are considered to be occluded.
  float occlusion_depth = (1 + sensor_noise_factor) * measurement_depth;
  if (kProtectSlightlyOccludedSurfels) {
    if (first_surfel_depth_value < occlusion_depth) {
      // TODO: Would it be better to use the front surfel's radius for that?
      occlusion_depth = (1 + kOcclusionDepthFactor) * first_surfel_depth_value;
    }
  }
  
  // Check if this surfel is occluded.
  if (cam_space_surfel_pos.z > occlusion_depth) {
    // Surfel is occluded.
    return false;
  }
  
  // The measurement supports the surfel. Determine whether they belong to the
  // same surface (then the measurement should be integrated into the surfel),
  // or to different surfaces (then the measurement must not be integrated).
  
  // Check whether the surfel normal looks towards the camera (instead of away from it).
  float surfel_distance = Norm(cam_space_surfel_pos);
  float3 global_surfel_normal = make_float3(surfels(kSurfelNormalX, surfel_index),
                                            surfels(kSurfelNormalY, surfel_index),
                                            surfels(kSurfelNormalZ, surfel_index));
  float3 local_surfel_normal = local_T_global.Rotate(global_surfel_normal);
  
  float dot_angle = (1.0f / surfel_distance) * (cam_space_surfel_pos.x * local_surfel_normal.x +
                                                cam_space_surfel_pos.y * local_surfel_normal.y +
                                                cam_space_surfel_pos.z * local_surfel_normal.z);
  if (dot_angle > kSurfelNormalToViewingDirThreshold) {
    return false;
  }
  
  // Check whether the surfel normal is compatible with the measurement normal.
  if (measurement_depth < cam_space_surfel_pos.z) {
    float2 normal = normals_buffer(y, x);
    float3 local_normal = make_float3(normal.x, normal.y, -sqrtf(::max(0.f, 1 - normal.x * normal.x - normal.y * normal.y)));
    
    float dot_angle = local_surfel_normal.x * local_normal.x +
                      local_surfel_normal.y * local_normal.y +
                      local_surfel_normal.z * local_normal.z;
    if (dot_angle < cos_normal_compatibility_threshold) {
      return false;
    }
  }
  
  // The measurement seems to belong to the same surface as the surfel.
  
  // Check whether the observation scale is compatible with the surfel scale.
  const float surfel_radius_squared = surfels(kSurfelRadiusSquared, surfel_index);
  if (kCheckScaleCompatibilityForIntegration) {
    const float observation_radius_squared = radius_buffer(y, x);
    if (observation_radius_squared / surfel_radius_squared > kMaxObservationRadiusFactorForIntegration * kMaxObservationRadiusFactorForIntegration) {
      return false;
    }
  }
  
  // Never merge the supported surfel.
  u32 supported_surfel = supporting_surfels(y, x);
  if (supported_surfel == surfel_index || supported_surfel == Surfel::kInvalidIndex) {
    return false;
  }
  
  // Compare the surfel to the supported surfel. Merge only if very similar.
  // Radius:
  const float other_radius_squared = surfels(kSurfelRadiusSquared, supported_surfel);
  float radius_diff = surfel_radius_squared / other_radius_squared;
  constexpr float kRadiusDiffThreshold = 1.2f;
  constexpr float kRadiusDiffThresholdSq = kRadiusDiffThreshold * kRadiusDiffThreshold;
  if (radius_diff > kRadiusDiffThresholdSq || radius_diff < 1 / kRadiusDiffThresholdSq) {
    return false;
  }
  
  // Distance:
  float3 other_global_position =
      make_float3(surfels(kSurfelX, supported_surfel),
                  surfels(kSurfelY, supported_surfel),
                  surfels(kSurfelZ, supported_surfel));
  float distance_squared = SquaredDistance(global_surfel_pos, other_global_position);
  constexpr float kDistanceThresholdFactor = 0.5f * (0.25f * 0.25f);
  if (distance_squared > kDistanceThresholdFactor * (surfel_radius_squared + other_radius_squared)) {
    return false;
  }
  
  // Normal:
  float3 other_surfel_normal = make_float3(surfels(kSurfelNormalX, supported_surfel),
                                           surfels(kSurfelNormalY, supported_surfel),
                                           surfels(kSurfelNormalZ, supported_surfel));
  dot_angle = Dot(global_surfel_normal, other_surfel_normal);
  constexpr float kCosNormalMergeThreshold = 0.93969f;  // 20 degrees
  if (dot_angle < kCosNormalMergeThreshold) {
    return false;
  }
  
  // Merge the surfel.
  *reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, surfel_index)) = 0;
  surfels(kSurfelRadiusSquared, surfel_index) = -1;
  *reinterpret_cast<u8*>(&reinterpret_cast<uchar4*>(&surfels(kSurfelColor, surfel_index))->w) = 1;  // Set neighbor detach request flag
  
  return true;
}

__global__ void MergeSurfelsCUDAKernel(
    u32 /*frame_index*/,
    int /*surfel_integration_active_window_size*/,
    float fx, float fy, float cx, float cy,
    CUDAMatrix3x4 local_T_global,
    float sensor_noise_factor,
    float cos_normal_compatibility_threshold,
    u32 surfel_count,
    CUDABuffer_<float> surfels,
    float depth_correction_factor,
    CUDABuffer_<u16> depth_buffer,
    CUDABuffer_<float2> normals_buffer,
    CUDABuffer_<float> radius_buffer,
    CUDABuffer_<u32> supporting_surfels,
    CUDABuffer_<u32> supporting_surfel_counts,
    CUDABuffer_<float> supporting_surfel_depth_sums,
    CUDABuffer_<u32> conflicting_surfels,
    CUDABuffer_<float> first_surfel_depth,
    CUDABuffer_<u32> num_merges_buffer) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  bool merged = false;
  
  if (surfel_index < surfel_count) {
//     if (IsSurfelActiveForIntegration(surfel_index, surfels, frame_index, surfel_integration_active_window_size)) {
    if (surfels(kSurfelRadiusSquared, surfel_index) >= 0) {
      float3 global_position =
          make_float3(surfels(kSurfelX, surfel_index),
                      surfels(kSurfelY, surfel_index),
                      surfels(kSurfelZ, surfel_index));
      float3 local_position = local_T_global * global_position;
      if (local_position.z > 0) {  // TODO: Compute z before x and y such that this early exit can be done earlier?
        float2 pixel_pos =
            make_float2(fx * (local_position.x / local_position.z) + cx,
                        fy * (local_position.y / local_position.z) + cy);
        int px = static_cast<int>(pixel_pos.x);
        int py = static_cast<int>(pixel_pos.y);
        if (!(pixel_pos.x < 0 || pixel_pos.y < 0 ||
            px < 0 || py < 0 ||
            px >= depth_buffer.width() || py >= depth_buffer.height())) {
          merged = ConsiderSurfelMergeAtPixel(
              px, py, local_position, global_position,
              surfel_index, surfels,
              local_T_global,
              sensor_noise_factor,
              cos_normal_compatibility_threshold,
              depth_correction_factor, depth_buffer, normals_buffer, radius_buffer, supporting_surfels,
              supporting_surfel_counts, supporting_surfel_depth_sums, conflicting_surfels, first_surfel_depth);
        }
      }
    }
  }
  
  typedef typename hipcub::BlockReduce<int, kMergeBlockWidth, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduceInt;
  __shared__ typename BlockReduceInt::TempStorage temp_storage;
  
  int num_merges = BlockReduceInt(temp_storage).Sum(merged ? 1 : 0);
  if (threadIdx.x == 0 && num_merges > 0) {
    atomicAdd(&num_merges_buffer(0, 0), static_cast<u32>(num_merges));
  }
}

void MergeSurfelsCUDA(
    hipStream_t stream,
    u32 frame_index,
    int surfel_integration_active_window_size,
    float sensor_noise_factor,
    float normal_compatibility_threshold_deg,
    const SE3f& local_T_global,
    const PinholeCamera4f& depth_camera,
    float depth_correction_factor,
    const CUDABuffer<u16>& depth_buffer,
    const CUDABuffer<float2>& normals_buffer,
    const CUDABuffer<float>& radius_buffer,
    CUDABuffer<u32>* supporting_surfels,
    CUDABuffer<u32>* supporting_surfel_counts,
    CUDABuffer<float>* supporting_surfel_depth_sums,
    CUDABuffer<u32>* conflicting_surfels,
    CUDABuffer<float>* first_surfel_depth,
    u32 surfel_count,
    u32* merge_count,
    CUDABuffer<float>* surfels) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  const float fx = depth_camera.parameters()[0];
  const float fy = depth_camera.parameters()[1];
  const float cx = depth_camera.parameters()[2];
  const float cy = depth_camera.parameters()[3];
  
  dim3 grid_dim(GetBlockCount(surfel_count, kMergeBlockWidth));
  dim3 block_dim(kMergeBlockWidth);
  
  static CUDABuffer<u32> num_merges_buffer(1, 1);  // TODO: do not use static
  num_merges_buffer.Clear(0, stream);
  
  MergeSurfelsCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      surfel_integration_active_window_size,
      fx, fy, cx, cy,
      CUDAMatrix3x4(local_T_global.matrix3x4()),
      sensor_noise_factor,
      cosf(M_PI / 180.0f * normal_compatibility_threshold_deg),
      surfel_count,
      surfels->ToCUDA(),
      depth_correction_factor,
      depth_buffer.ToCUDA(),
      normals_buffer.ToCUDA(),
      radius_buffer.ToCUDA(),
      supporting_surfels->ToCUDA(),
      supporting_surfel_counts->ToCUDA(),
      supporting_surfel_depth_sums->ToCUDA(),
      conflicting_surfels->ToCUDA(),
      first_surfel_depth->ToCUDA(),
      num_merges_buffer.ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  u32 num_merges = 0;
  num_merges_buffer.DownloadAsync(stream, &num_merges);
  hipStreamSynchronize(stream);
  *merge_count += num_merges;
}


__global__ void RegularizeSurfelsCUDAClearGradientsKernel(
    u32 surfel_count,
    CUDABuffer_<float> surfels) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    // TODO: Put this in the last kernel of the denoising (and in the
    //       initialization) and expect that it remains zero in-between the
    //       calls in order to save one kernel call? Is this used anywhere else?
    surfels(kSurfelGradientX, surfel_index) = 0;
    surfels(kSurfelGradientY, surfel_index) = 0;
    surfels(kSurfelGradientZ, surfel_index) = 0;
    surfels(kSurfelGradientCount, surfel_index) = 0;
  }
}

__global__ void RegularizeSurfelsCUDAAccumulateNeighborGradientsKernel(
    u32 frame_index,
    int regularization_frame_window_size,
    float radius_factor_for_regularization_neighbors_squared,
    float regularizer_weight,
    u32 surfel_count,
    CUDABuffer_<float> surfels) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    // Count neighbors.
    int neighbor_count = 0;
    for (int neighbor_index = 0; neighbor_index < kSurfelNeighborCount; ++ neighbor_index) {
      u32 neighbor_surfel_index = *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + neighbor_index, surfel_index));
      if (neighbor_surfel_index == Surfel::kInvalidIndex) {
        continue;
      }
      if (static_cast<int>(*reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, neighbor_surfel_index))) < static_cast<int>(frame_index - regularization_frame_window_size)) {
        continue;
      }
      ++ neighbor_count;
    }
    if (neighbor_count == 0) {
      return;
    }
    
    float3 smooth_position =
        make_float3(surfels(kSurfelSmoothX, surfel_index),
                    surfels(kSurfelSmoothY, surfel_index),
                    surfels(kSurfelSmoothZ, surfel_index));
    float3 normal =
        make_float3(surfels(kSurfelNormalX, surfel_index),
                    surfels(kSurfelNormalY, surfel_index),
                    surfels(kSurfelNormalZ, surfel_index));
    
    const float surfel_radius_squared = surfels(kSurfelRadiusSquared, surfel_index);
    
    // Accumulate gradient terms for neighbors.
    float factor = 2 * regularizer_weight / neighbor_count;
    for (int neighbor_index = 0; neighbor_index < kSurfelNeighborCount; ++ neighbor_index) {
      u32 neighbor_surfel_index = *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + neighbor_index, surfel_index));
      if (neighbor_surfel_index == Surfel::kInvalidIndex) {
        continue;
      }
      if (static_cast<int>(*reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, neighbor_surfel_index))) < static_cast<int>(frame_index - regularization_frame_window_size)) {
        continue;
      }
      
      float3 neighbor_position =
          make_float3(surfels(kSurfelSmoothX, neighbor_surfel_index),
                      surfels(kSurfelSmoothY, neighbor_surfel_index),
                      surfels(kSurfelSmoothZ, neighbor_surfel_index));
      
      float3 this_to_neighbor =
          make_float3(neighbor_position.x - smooth_position.x,
                      neighbor_position.y - smooth_position.y,
                      neighbor_position.z - smooth_position.z);
      float factor_times_normal_dot_difference = factor * (normal.x * this_to_neighbor.x + normal.y * this_to_neighbor.y + normal.z * this_to_neighbor.z);
      
      float3 gradient_term_for_neighbor =
          make_float3(factor_times_normal_dot_difference * normal.x,
                      factor_times_normal_dot_difference * normal.y,
                      factor_times_normal_dot_difference * normal.z);
      
      atomicAdd(&surfels(kSurfelGradientX, neighbor_surfel_index), gradient_term_for_neighbor.x);
      atomicAdd(&surfels(kSurfelGradientY, neighbor_surfel_index), gradient_term_for_neighbor.y);
      atomicAdd(&surfels(kSurfelGradientZ, neighbor_surfel_index), gradient_term_for_neighbor.z);
      atomicAdd(&surfels(kSurfelGradientCount, neighbor_surfel_index), regularizer_weight / neighbor_count);
      
      // If the neighbor is too far away, remove it.
      // NOTE / TODO: it can still happen that there are far away but inactive
      //              neighbors, which will influence an active surfel, since
      //              this check only removes active neighbors.
      //              However, I think this should be relatively rare.
      float neighbor_distance_squared = this_to_neighbor.x * this_to_neighbor.x + this_to_neighbor.y * this_to_neighbor.y + this_to_neighbor.z * this_to_neighbor.z;
      if (neighbor_distance_squared > radius_factor_for_regularization_neighbors_squared * surfel_radius_squared) {
        *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + neighbor_index, surfel_index)) = Surfel::kInvalidIndex;
      }
    }
  }
}

__global__ void RegularizeSurfelsCUDAKernel(
    u32 frame_index,
    int regularization_frame_window_size,
    float regularizer_weight,
    u32 surfel_count,
    CUDABuffer_<float> surfels) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    if (static_cast<int>(*reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, surfel_index))) < static_cast<int>(frame_index - regularization_frame_window_size)) {
      return;
    }
    
    float3 measured_position =
        make_float3(surfels(kSurfelX, surfel_index),
                    surfels(kSurfelY, surfel_index),
                    surfels(kSurfelZ, surfel_index));
    float3 smooth_position =
        make_float3(surfels(kSurfelSmoothX, surfel_index),
                    surfels(kSurfelSmoothY, surfel_index),
                    surfels(kSurfelSmoothZ, surfel_index));
    float3 normal =
        make_float3(surfels(kSurfelNormalX, surfel_index),
                    surfels(kSurfelNormalY, surfel_index),
                    surfels(kSurfelNormalZ, surfel_index));
    
    // Data term and neighbor-induced gradient terms
    constexpr float data_term_factor = 2;
    float3 gradient =
        make_float3(data_term_factor * (smooth_position.x - measured_position.x) + surfels(kSurfelGradientX, surfel_index),
                    data_term_factor * (smooth_position.y - measured_position.y) + surfels(kSurfelGradientY, surfel_index),
                    data_term_factor * (smooth_position.z - measured_position.z) + surfels(kSurfelGradientZ, surfel_index));
    
    // Regularization gradient terms
    int neighbor_count = 0;
    float3 regularization_gradient = make_float3(0, 0, 0);
    for (int neighbor_index = 0; neighbor_index < kSurfelNeighborCount; ++ neighbor_index) {
      u32 neighbor_surfel_index = *reinterpret_cast<u32*>(&surfels(kSurfelNeighbor0 + neighbor_index, surfel_index));
      if (neighbor_surfel_index == Surfel::kInvalidIndex) {
        continue;
      }
      
      ++ neighbor_count;
      
      float3 neighbor_position =
          make_float3(surfels(kSurfelSmoothX, neighbor_surfel_index),
                      surfels(kSurfelSmoothY, neighbor_surfel_index),
                      surfels(kSurfelSmoothZ, neighbor_surfel_index));
      
      float3 this_to_neighbor =
          make_float3(neighbor_position.x - smooth_position.x,
                      neighbor_position.y - smooth_position.y,
                      neighbor_position.z - smooth_position.z);
      float normal_dot_difference = normal.x * this_to_neighbor.x + normal.y * this_to_neighbor.y + normal.z * this_to_neighbor.z;
      
      regularization_gradient =
          make_float3(regularization_gradient.x - normal_dot_difference * normal.x,
                      regularization_gradient.y - normal_dot_difference * normal.y,
                      regularization_gradient.z - normal_dot_difference * normal.z);
    }
    
    if (neighbor_count > 0) {
      // Apply constant factor to regularization gradient term
      float factor = 2 * regularizer_weight / neighbor_count;
      gradient =
          make_float3(gradient.x + factor * regularization_gradient.x,
                      gradient.y + factor * regularization_gradient.y,
                      gradient.z + factor * regularization_gradient.z);
    }
    
    const float residual_terms_weight_sum = 1 + regularizer_weight + surfels(kSurfelGradientCount, surfel_index);
    const float kStepSizeFactor = 0.5f / residual_terms_weight_sum;
    
    // Avoid divergence by limiting the step length to a multiple of the surfel
    // radius (multiple with this factor here).
    // TODO: It seems that this is not necessary anymore now that the step size
    //       is more intelligently chosen. Remove it (after some more extensive
    //       testing).
    constexpr float kMaxStepLengthFactor = 1.0f;
    float max_step_length = kMaxStepLengthFactor * sqrtf(surfels(kSurfelRadiusSquared, surfel_index));
    float step_length = kStepSizeFactor * sqrtf(gradient.x * gradient.x + gradient.y * gradient.y + gradient.z * gradient.z);
    float step_factor = kStepSizeFactor;
    if (step_length > max_step_length) {
      step_factor = max_step_length / step_length * kStepSizeFactor;
    }
    
    // NOTE: Writing the update into the gradient first to avoid race conditions
    //       (the smooth position may still be used by neighboring surfel updates).
    //       The next kernel call will move the result to the smooth position field.
    surfels(kSurfelGradientX, surfel_index) = smooth_position.x - step_factor * gradient.x;
    surfels(kSurfelGradientY, surfel_index) = smooth_position.y - step_factor * gradient.y;
    surfels(kSurfelGradientZ, surfel_index) = smooth_position.z - step_factor * gradient.z;
  }
}

__global__ void RegularizeSurfelsCUDAUpdateKernel(
    u32 frame_index,
    int regularization_frame_window_size,
    u32 surfel_count,
    CUDABuffer_<float> surfels) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    if (static_cast<int>(*reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, surfel_index))) < static_cast<int>(frame_index - regularization_frame_window_size)) {
      return;
    }
    
    surfels(kSurfelSmoothX, surfel_index) = surfels(kSurfelGradientX, surfel_index);
    surfels(kSurfelSmoothY, surfel_index) = surfels(kSurfelGradientY, surfel_index);
    surfels(kSurfelSmoothZ, surfel_index) = surfels(kSurfelGradientZ, surfel_index);
  }
}

__global__ void RegularizeSurfelsCUDACopyOnlyKernel(
    u32 frame_index,
    int regularization_frame_window_size,
    u32 surfel_count,
    CUDABuffer_<float> surfels) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    // TODO: Only changed surfels need to be touched here.
    if (static_cast<int>(*reinterpret_cast<u32*>(&surfels(kSurfelLastUpdateStamp, surfel_index))) < static_cast<int>(frame_index - regularization_frame_window_size)) {
      return;
    }
    
    surfels(kSurfelSmoothX, surfel_index) = surfels(kSurfelX, surfel_index);
    surfels(kSurfelSmoothY, surfel_index) = surfels(kSurfelY, surfel_index);
    surfels(kSurfelSmoothZ, surfel_index) = surfels(kSurfelZ, surfel_index);
  }
}

void RegularizeSurfelsCUDA(
    hipStream_t stream,
    bool disable_denoising,
    u32 frame_index,
    float radius_factor_for_regularization_neighbors,
    float regularizer_weight,
    int regularization_frame_window_size,
    u32 surfel_count,
    CUDABuffer<float>* surfels) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  constexpr int kBlockWidth = 1024;
  dim3 grid_dim(GetBlockCount(surfel_count, kBlockWidth));
  dim3 block_dim(kBlockWidth);
  
  if (disable_denoising) {
    // Only copy the raw surfel positions to the smoothed position fields.
    RegularizeSurfelsCUDACopyOnlyKernel
    <<<grid_dim, block_dim, 0, stream>>>(
        frame_index,
        regularization_frame_window_size,
        surfel_count,
        surfels->ToCUDA());
    #ifdef CUDA_SEQUENTIAL_CHECKS
      hipDeviceSynchronize();
    #endif
    CHECK_CUDA_NO_ERROR();
    return;
  }
  
  RegularizeSurfelsCUDAClearGradientsKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      surfel_count,
      surfels->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  RegularizeSurfelsCUDAAccumulateNeighborGradientsKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      regularization_frame_window_size,
      radius_factor_for_regularization_neighbors * radius_factor_for_regularization_neighbors,
      regularizer_weight,
      surfel_count,
      surfels->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  RegularizeSurfelsCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      regularization_frame_window_size,
      regularizer_weight,
      surfel_count,
      surfels->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  RegularizeSurfelsCUDAUpdateKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      frame_index,
      regularization_frame_window_size,
      surfel_count,
      surfels->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
}

__global__ void ExportVerticesCUDAKernel(
    u32 surfel_count,
    CUDABuffer_<float> surfels,
    CUDABuffer_<float> position_buffer,
    CUDABuffer_<u8> color_buffer) {
  unsigned int surfel_index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (surfel_index < surfel_count) {
    bool merged = surfels(kSurfelRadiusSquared, surfel_index) < 0;
    
    float* position_ptr = position_buffer.address();
    position_ptr[3 * surfel_index + 0] = merged ? HIP_NAN_F : surfels(kSurfelSmoothX, surfel_index);
    position_ptr[3 * surfel_index + 1] = merged ? HIP_NAN_F : surfels(kSurfelSmoothY, surfel_index);
    position_ptr[3 * surfel_index + 2] = merged ? HIP_NAN_F : surfels(kSurfelSmoothZ, surfel_index);
    
    const uchar4 color = *(reinterpret_cast<uchar4*>(&surfels(kSurfelColor, surfel_index)));
    u8* color_ptr = color_buffer.address();
    color_ptr[3 * surfel_index + 0] = color.x;
    color_ptr[3 * surfel_index + 1] = color.y;
    color_ptr[3 * surfel_index + 2] = color.z;
  }
}

void ExportVerticesCUDA(
    hipStream_t stream,
    u32 surfel_count,
    const CUDABuffer<float>& surfels,
    CUDABuffer<float>* position_buffer,
    CUDABuffer<u8>* color_buffer) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  if (surfel_count == 0) {
    return;
  }
  
  constexpr int kBlockWidth = 1024;
  dim3 grid_dim(GetBlockCount(surfel_count, kBlockWidth));
  dim3 block_dim(kBlockWidth);
  
  ExportVerticesCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      surfel_count,
      surfels.ToCUDA(),
      position_buffer->ToCUDA(),
      color_buffer->ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
}

__global__ void DebugPrintSurfelCUDAKernel(
    usize surfel_index,
    CUDABuffer_<float> surfels) {
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    printf("DEBUGGING surfel %i on GPU ...\n", static_cast<int>(surfel_index));
    
    printf("DEBUG surfel raw position x on GPU: %f\n", surfels(kSurfelX, surfel_index));
    printf("DEBUG surfel raw position y on GPU: %f\n", surfels(kSurfelY, surfel_index));
    printf("DEBUG surfel raw position z on GPU: %f\n", surfels(kSurfelZ, surfel_index));
    
    printf("DEBUG surfel smooth position x on GPU: %f\n", surfels(kSurfelSmoothX, surfel_index));
    printf("DEBUG surfel smooth position y on GPU: %f\n", surfels(kSurfelSmoothY, surfel_index));
    printf("DEBUG surfel smooth position z on GPU: %f\n", surfels(kSurfelSmoothZ, surfel_index));
    
    printf("DEBUG surfel creation stamp on GPU: %i\n", static_cast<int>(*reinterpret_cast<u32*>(&surfels(kSurfelCreationStamp, surfel_index))));
  }
}

void DebugPrintSurfelCUDA(
    hipStream_t stream,
    usize surfel_index,
    const CUDABuffer<float>& surfels) {
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
  
  constexpr int kBlockWidth = 32;
  dim3 grid_dim(1);
  dim3 block_dim(kBlockWidth);
  DebugPrintSurfelCUDAKernel
  <<<grid_dim, block_dim, 0, stream>>>(
      surfel_index,
      surfels.ToCUDA());
  #ifdef CUDA_SEQUENTIAL_CHECKS
    hipDeviceSynchronize();
  #endif
  CHECK_CUDA_NO_ERROR();
}

}
